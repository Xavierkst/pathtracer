#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include "Geometries.h"
#include "Payloads.h"


using namespace optix;

rtBuffer<Sphere> spheres; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

// pass to payload: hitPt, Normal at hitPt
rtDeclareVariable(Payload, payload, rtPayload, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and sphere
    Sphere sphere = spheres[primIndex];
    float t, t_sol1, t_sol2;
    float a, b, c;
    // TODO: implement sphere intersection test here
    // there are 4 outcomes for the ray here
    // completely miss. ray pass intersect 2 pts. 
    // ray skims the surface. 
    // ray intersects only 1 pt, i.e. ray starts from inside

    // apply the inverse transform to ray here before calculating 
    // sphere intersection
    float3 ray_orig = make_float3(sphere.transform.inverse() * make_float4(ray.origin, 1));
    float3 ray_dir = normalize(make_float3(sphere.transform.inverse() * make_float4(ray.direction, 0)));

    // use quadratic equation and find solutions for t 
    optix::float3 rad = ray_orig - sphere.position; 
    a = dot(ray_dir, ray_dir); 
    b = 2.0f * dot(ray_dir, rad);
    c = dot(rad, rad) - (sphere.radius * sphere.radius);

    // find discriminant i.e. the value inside sqrt (b^2 - 4ac) 
    float disc = (b * b) - (4 * a * c);

    // if discrim is negative, no intersection
    if (disc < .0f) return; 

    disc = sqrtf(disc); 
    // both values are legal
    t_sol1 = (-1.0f * b + disc ) / (2.0f * a);
    t_sol2 = (-1.0f * b - disc ) / (2.0f * a);
    
    // find the smaller of the 2
    if (t_sol2 < t_sol1) {
        float t_temp = t_sol1;
        t_sol1 = t_sol2;
        t_sol2 = t_temp;
    }
    // now check to make sure we chose a positive param-distance
    if (t_sol1 < .0f) t_sol1 = t_sol2;
    // both solutions are negative
    if (t_sol1 < .0f) return; 

    t = t_sol1;

    // find normal at hitPoint, pass into payload
    float3 hitPt = ray_orig + t * ray_dir; 
    float3 hitPtNormal = normalize(hitPt - sphere.position);

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // TODO: assign attribute variables here
        // Pass attributes
        attrib = sphere.attributes;
        //rtPrintf("made it! %f", hitPt.x);
        // Pass hitPt and normal at hitPt into payload 
        // to calculate payload.radiance in closestHit()

        // transform hitPt back: Mp 
        payload.hitPoint = 
            make_float3(sphere.transform * make_float4(hitPt, 1)); // applying Mp 
        // transform normal at hitPoint back (M-1)^T
        payload.hitPointNormal = normalize(
            make_float3((sphere.transform.inverse()).transpose() * 
                make_float4(hitPtNormal, 0)));

        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Sphere sphere = spheres[primIndex];

    // TODO: implement sphere bouding box
    result[0] = -1000.f;
    result[1] = -1000.f;
    result[2] = -1000.f;
    result[3] = 1000.f;
    result[4] = 1000.f;
    result[5] = 1000.f;
}