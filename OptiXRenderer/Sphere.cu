#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include "Geometries.h"
#include "Payloads.h"


using namespace optix;

rtBuffer<Sphere> spheres; // a buffer of all spheres

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );
rtDeclareVariable(intersectionData, intersectData, attribute intersectData, );

// pass to payload: hitPt, Normal at hitPt
rtDeclareVariable(Payload, payload, rtPayload, );
//rtDeclareVariable(ShadowPayload, shadowPayload, rtPayload, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and sphere
    Sphere sphere = spheres[primIndex];
    float t, t_sol1, t_sol2;
    float a, b, c;
    float epsilon = 0.001f;

    // TODO: implement sphere intersection test here
    // there are 4 outcomes for the ray here
    // completely miss. ray pass intersect 2 pts. 
    // ray skims the surface. 
    // ray intersects only 1 pt, i.e. ray starts from inside

    // apply the inverse transform to ray here before calculating 
    // sphere intersection
    float4 temp_ray_orig = sphere.transform.inverse() * make_float4(ray.origin, 1);
    float3 ray_orig = make_float3(temp_ray_orig / (float)temp_ray_orig.w);
    float3 ray_dir = normalize(make_float3(sphere.transform.inverse() * make_float4(ray.direction, 0)));

    // use quadratic equation and find solutions for t 
    optix::float3 rad = ray_orig - sphere.position; 
    a = dot(ray_dir, ray_dir); 
    b = 2.0f * dot(ray_dir, rad);
    c = dot(rad, rad) - (sphere.radius * sphere.radius);

    // find discriminant i.e. the value inside sqrt (b^2 - 4ac) 
    float disc = (b * b) - (4 * a * c);

    // if discrim is negative, no intersection
    if (disc < .0f) return; 

    disc = sqrtf(disc); 
    // both values are legal
    t_sol1 = (-1.0f * b + disc ) / (2.0f * a);
    t_sol2 = (-1.0f * b - disc ) / (2.0f * a);
    
    // find the smaller of the 2
    if (t_sol2 < t_sol1) {
        float t_temp = t_sol1;
        t_sol1 = t_sol2;
        t_sol2 = t_temp;
    }
    // now check to make sure we chose a positive param-distance
    if (t_sol1 < .0f) t_sol1 = t_sol2;
    // both solutions are negative
    if (t_sol1 < .0f) return; 

    t = t_sol1;

    // find normal at hitPoint, pass into payload
    float3 hitPt = ray_orig + t * ray_dir; 

    // obtain normal
    //float3 hitPtNormal = normalize( hitPt - make_float3(sphere.transform.inverse() * make_float4(sphere.position, 1.0f)));
    float3 hitPtNormal = normalize(hitPt - sphere.position);
    //hitPt += epsilon * hitPtNormal;
    // transform normal to worldspace
    hitPtNormal = normalize(make_float3(
        (sphere.transform.inverse()).transpose() * make_float4(hitPtNormal, 0)));

    // transform hitPoint back into worldspace from local space
    float4 temp_pt = sphere.transform * make_float4(hitPt, 1);
    hitPt = make_float3(temp_pt / (float)temp_pt.w)/* + epsilon * hitPtNormal*/;/*+epsilon * hitPtNormal;*/

    // find distance t:  
    t = length(hitPt - ray.origin);
    
    // Account for shadow Acne??  
    //hitPt += hitPtNormal * epsilon;

    // compute reflection ray direction in world space
    //float3 reflectionDir = normalize(ray_dir - 2.0f * dot(ray_dir, hitPtNormal) * hitPtNormal);
    float3 reflectionDir = normalize(ray.direction - (2.0f * dot( ray.direction, hitPtNormal) * hitPtNormal));
    //rtPrintf("%f, %f, %f", reflectionDir.x, reflectionDir.y, reflectionDir.z);

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // TODO: assign attribute variables here
        // Pass attributes
        attrib = sphere.attributes;       
        //rtPrintf("sphere spec: %f %f %f\n", attrib.specular.x, attrib.specular.y, attrib.specular.z);
        intersectData.hitPoint = hitPt;
        
        intersectData.hitPointNormal = hitPtNormal;
        intersectData.reflectDir = reflectionDir;
        intersectData.rayDir = ray.direction;
        intersectData.rayOrig = ray.origin;
        //rtPrintf("made it! %f", hitPt.x);
        // Pass hitPt and normal at hitPt into payload 
        // to calculate payload.radiance in closestHit()

        //payload.hitPoint = hitPt; 
        //    //make_float3(sphere.transform * make_float4(hitPt, 1)); // applying Mp 
        //// transform normal at hitPoint back (M-1)^T
        //payload.hitPointNormal = hitPtNormal;
        //payload.dir = reflectionDir;

        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Sphere sphere = spheres[primIndex];

    // TODO: implement sphere bouding box
    result[0] = -1000.f;
    result[1] = -1000.f;
    result[2] = -1000.f;
    result[3] = 1000.f;
    result[4] = 1000.f;
    result[5] = 1000.f;

    //// as explained in: 
    //// https://tavianator.com/2014/ellipsoid_bounding_boxes.html

    //Matrix4x4 S = Matrix4x4::identity(); 
    //float rad_2 = (sphere.radius);
    //S.setRow(0, make_float4(rad_2, .0f, .0f, .0f));
    //S.setRow(1, make_float4(.0f, rad_2, .0f, .0f));
    //S.setRow(2, make_float4(.0f, .0f, rad_2, .0f));
    //S.setRow(3, make_float4(.0f, .0f, .0f, -1.0f));
    ////S[0] = rad_2;
    ////S[5] = rad_2;
    ////S[10] = rad_2;
    ////S[15] = -1.0f;
    //Matrix4x4 M = sphere.transform;
    //////Matrix4x4 Q = M.inverse().transpose() * S * M.inverse();
    ////Matrix4x4 M_T = M.transpose();
    ////Matrix4x4 R = M * S.inverse() * M_T;
    //////Matrix4x4 R = Q.inverse();

    //////if (R == R.transpose())
    //////    rtPrintf("same!");
    //float xMax, xMin, yMax, yMin, zMax, zMin;

    //////xMax = (R[3] + sqrtf(powf(R[3], 2.0f) - (R[15] * R[0]))) / (float)R[15];
    //////xMin = (R[3] - sqrtf(powf(R[3], 2.0f) - (R[15] * R[0]))) / (float)R[15];
    //////yMax = (R[7] + sqrtf(powf(R[7], 2.0f) - (R[15] * R[5]))) / (float)R[15];
    //////yMin = (R[7] - sqrtf(powf(R[7], 2.0f) - (R[15] * R[5]))) / (float)R[15];
    //////zMax = (R[11] + sqrtf(powf(R[11], 2.0f) - (R[10] * R[15]))) / (float)R[15];
    //////zMin = (R[11] - sqrtf(powf(R[11], 2.0f) - (R[10] * R[15]))) / (float)R[15];

    //// we end up only requiring the matrix M in the calculation
    //// after plugging R[i][j] into the plane equations, where 
    //// S is 4x4 mat representing sphere (rad and position), and
    //// R == Q.inverse == M_T.inverse * S * M.inverse. Hence:

    //// eg. xmin or xmax = M[1][4] +/- sqrt( M[1][1]^2 + M[1][2]^2 + M[1][3]^2 ) 
    //xMax = M[3] + sqrtf( powf(M[0], 2.0f) + powf(M[1], 2.0f) + powf(M[2], 2.0f));
    //xMin = M[3] - sqrtf( powf(M[0], 2.0f) + powf(M[1], 2.0f) + powf(M[2], 2.0f));
    //yMax = M[7] + sqrtf( powf(M[4], 2.0f) + powf(M[5], 2.0f) + powf(M[6], 2.0f));
    //yMin = M[7] - sqrtf( powf(M[4], 2.0f) + powf(M[5], 2.0f) + powf(M[6], 2.0f));
    //zMax = M[11] + sqrtf( powf(M[8], 2.0f) + powf(M[9], 2.0f) + powf(M[10], 2.0f));
    //zMin = M[11] - sqrtf( powf(M[8], 2.0f) + powf(M[9], 2.0f) + powf(M[10], 2.0f));

    //result[0] = xMin;
    //result[1] = yMin;
    //result[2] = zMin;
    //result[3] = xMax;
    //result[4] = yMax;
    //result[5] = zMax;
    float x, y, z;    
    x = length(make_float3(sphere.transform.getRow(0)));
    y = length(make_float3(sphere.transform.getRow(1)));
    z = length(make_float3(sphere.transform.getRow(2)));
    result[0] = sphere.transform[3] - x;
    result[1] = sphere.transform[7] - y;
    result[2] = sphere.transform[11] - z;
    result[3] = sphere.transform[3] + x;
    result[4] = sphere.transform[7] + y;
    result[5] = sphere.transform[11] + z;
}