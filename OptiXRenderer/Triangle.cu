#include "hip/hip_runtime.h"
#include "optix.h"
#include "optix_device.h"
#include "Geometries.h"
#include "Payloads.h"

using namespace optix;

rtBuffer<Triangle> triangles; // a buffer of all triangles 

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );
rtDeclareVariable(intersectionData, intersectData, attribute intersectData, );

// Transfer values into payload variable for color calc in closestHit()
rtDeclareVariable(Payload, payload, rtPayload, );
//rtDeclareVariable(ShadowPayload, shadowPayload, rtPayload, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and triangle
    Triangle tri = triangles[primIndex];
    float t;
    float epsilon = 0.001f;

    // TODO: implement triangle intersection test here
    
    // get plane normal
    float3 edge1 = tri.vertices[1] - tri.vertices[0]; 
    float3 edge2 = tri.vertices[2] - tri.vertices[0]; 
    // we have normal of triangle, N
    float3 N = normalize(cross(edge1, edge2)); 

    // triangles don't need to worry about transforming
    // ray by M-1 since it is still a triangle (but for spheres, 
    // must consider)
    float4 temp_ray = tri.transform.inverse() * make_float4(ray.origin, 1);
    float3 ray_orig = make_float3(temp_ray / (float)temp_ray.w);
    //ray_orig = make_float3(ray)
    float3 ray_dir = normalize(make_float3(tri.transform.inverse() * make_float4(ray.direction, 0)));

    //float3 ray_orig = ray.origin;
    //float3 ray_dir = normalize(ray.direction);
    // find parametric dist t: 
    //t = (dot(tri.vertices[0], N) - dot(ray.origin, N)) / dot(ray.direction, N);
    t = (dot(tri.vertices[0], N) - dot(ray_orig, N)) / dot(ray_dir, N);
    if (t < 0) {
        // triangle is not positive distance to ray, i.e. behind
        return; 
    }

    // check if ray is within the triangle, use barycentric
    // we use cross prod of an edge in the tri and the hit point
    // we know hit point is outside of triangle if dot product of 
    // Normal and orthogonal vect is < 0
    // Note: tri is ACW 
    float3 hitPt = ray_orig + t * ray_dir /*+ N*epsilon*/; // account for shadow acne: 
    
    float3 orthogEdge;
    // check 1 (total 3 edges to check) 
    float3 edgeV1V0 = tri.vertices[1] - tri.vertices[0]; 
    float3 edgePV0 = hitPt - tri.vertices[0]; 
    orthogEdge = cross(edgeV1V0, edgePV0); 
    // calc u, v or w here: 

    // check if hitPt inside or outside tri 
    if (dot(N, orthogEdge) < 0)
        return; // hitPt outside

    // Check next edge
    float3 edgeV2V1 = tri.vertices[2] - tri.vertices[1];
    float3 edgePV1 = hitPt - tri.vertices[1];
    orthogEdge = cross(edgeV2V1, edgePV1);

    if (dot(N, orthogEdge) < 0)
        return; 

    // last check: edgeV0V2 and edgePV2
    float3 edgeV0V2 = tri.vertices[0] - tri.vertices[2]; 
    float3 edgePV2 = hitPt - tri.vertices[2];
    orthogEdge = cross(edgeV0V2, edgePV2);

    if (dot(N, orthogEdge) < 0)
        return;

    // made it here, means the hit point is inside the tri!

    // find normal at hitPoint, pass into payload
    // Note: the triangle already transformed during parsing
    //float3 hitPt = ray.origin + t * ray.direction; 
     //cross prod of any 2 edges from above


    // transform normal to worldspace
    float3 hitPtNormal = normalize(make_float3((
        tri.transform.inverse()).transpose() * make_float4(N, 0)));
    // transform hit point to worldspace
    float4 temp_hit = tri.transform * make_float4(hitPt, 1);
    hitPt = make_float3(temp_hit / (float) temp_hit.w) /*+ epsilon * hitPtNormal*/;

    //// obtain parametric distance to hitPoint in worldspace
    t = length(hitPt - ray.origin);

    // compute reflection ray direction
    float3 reflectionDir = normalize(ray.direction - (2.0f * dot(ray.direction, hitPtNormal) * hitPtNormal));
    //float3 reflectionDir = normalize(ray.direction - 2.0f * dot(ray.direction, N) * N);

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // TODO: assign attribute variables here
        // Pass attributes: i.e. materials of the object
        attrib = tri.attributes;
        intersectData.hitPoint = hitPt;
        intersectData.hitPointNormal = /*hitPtNormal*/N;
        intersectData.reflectDir = reflectionDir;
        intersectData.rayDir = ray.direction;
        intersectData.rayOrig = ray.origin;
        // Pass hitPt and normal at hitPt into payload 
        // to calculate payload.radiance in closestHit()
        //payload.hitPoint = hitPt;
        //payload.hitPointNormal = hitPtNormal;
        //payload.dir = reflectionDir;

        //shadowPayload.isVisible = true;

        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Triangle tri = triangles[primIndex];

    result[0] = -1000.f;
    result[1] = -1000.f;
    result[2] = -1000.f;
    result[3] = 1000.f;
    result[4] = 1000.f;
    result[5] = 1000.f;

    // TODO: implement triangle bouding box
    if (tri.vertices[0].x > result[0]) {
        result[0] = tri.vertices[0].x;
    }
    
    if (tri.vertices[1].x > result[0]) {
        result[0] = tri.vertices[1].x;
    }

    if (tri.vertices[2].x > result[0]) {
        result[0] = tri.vertices[2].x;
    }

    // y 
    if (tri.vertices[0].y > result[0]) {
        result[0] = tri.vertices[0].y;
    }
    
    if (tri.vertices[1].y > result[0]) {
        result[0] = tri.vertices[1].y;
    }

    if (tri.vertices[2].y > result[0]) {
        result[0] = tri.vertices[2].y;
    }
    
    // z
    if (tri.vertices[0].z > result[0]) {
        result[0] = tri.vertices[0].z;
    }
    
    if (tri.vertices[1].z > result[0]) {
        result[0] = tri.vertices[1].z;
    }

    if (tri.vertices[2].z > result[0]) {
        result[0] = tri.vertices[2].z;
    }

}