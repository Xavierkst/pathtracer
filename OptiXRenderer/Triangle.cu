#include "hip/hip_runtime.h"
#include "optix.h"
#include "optix_device.h"
#include "Geometries.h"
#include "Payloads.h"

using namespace optix;

rtBuffer<Triangle> triangles; // a buffer of all triangles 

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );
rtDeclareVariable(intersectionData, intersectData, attribute intersectData, );

// Transfer values into payload variable for color calc in closestHit()
rtDeclareVariable(Payload, payload, rtPayload, );
//rtDeclareVariable(ShadowPayload, shadowPayload, rtPayload, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and triangle
    Triangle tri = triangles[primIndex];
    float t;
    float epsilon = 0.001f;

    // TODO: implement triangle intersection test here
    
    // get plane normal
    float3 edge1 = tri.vertices[1] - tri.vertices[0]; 
    float3 edge2 = tri.vertices[2] - tri.vertices[0]; 
    // we have normal of triangle, N
    float3 N = normalize(cross(edge1, edge2)); 

    // triangles don't need to worry about transforming
    // ray by M-1 since it is still a triangle (but for spheres, 
    // must consider)
    //float4 temp_ray = tri.transform.inverse() * make_float4(ray.origin, 1);
    //float3 ray_orig = make_float3(temp_ray / (float)temp_ray.w);
    ////ray_orig = make_float3(ray)
    //float3 ray_dir = normalize(make_float3(tri.transform.inverse() * make_float4(ray.direction, 0)));

    float3 ray_orig = ray.origin;
    float3 ray_dir = normalize(ray.direction);
    // find parametric dist t: 
    //t = (dot(tri.vertices[0], N) - dot(ray.origin, N)) / dot(ray.direction, N);
    float parallel = dot(ray_dir, N);
    if (parallel == .0f) return; 
    t = (dot(tri.vertices[0], N) - dot(ray_orig, N)) / parallel;
    if (t < 0) {
        // triangle is not positive distance to ray, i.e. behind
        return; 
    }

    // check if ray is within the triangle, use barycentric
    // we use cross prod of an edge in the tri and the hit point
    // we know hit point is outside of triangle if dot product of 
    // Normal and orthogonal vect is < 0
    // Note: tri is ACW 
    float3 hitPt = ray_orig + t * ray_dir /*+ N*epsilon*/; // account for shadow acne: 
    
    float3 orthogEdge;
    // check 1 (total 3 edges to check) 
    float3 edgeV1V0 = tri.vertices[1] - tri.vertices[0]; 
    float3 edgePV0 = hitPt - tri.vertices[0]; 
    orthogEdge = cross(edgeV1V0, edgePV0); 
    // calc u, v or w here: 

    // check if hitPt inside or outside tri 
    if (dot(N, orthogEdge) < 0)
        return; // hitPt outside

    // Check next edge
    float3 edgeV2V1 = tri.vertices[2] - tri.vertices[1];
    float3 edgePV1 = hitPt - tri.vertices[1];
    orthogEdge = cross(edgeV2V1, edgePV1);

    if (dot(N, orthogEdge) < 0)
        return; 

    // last check: edgeV0V2 and edgePV2
    float3 edgeV0V2 = tri.vertices[0] - tri.vertices[2]; 
    float3 edgePV2 = hitPt - tri.vertices[2];
    orthogEdge = cross(edgeV0V2, edgePV2);

    if (dot(N, orthogEdge) < 0)
        return;

    // made it here, means the hit point is inside the tri!

    // find normal at hitPoint, pass into payload
    // Note: the triangle already transformed during parsing
    //float3 hitPt = ray.origin + t * ray.direction; 
     //cross prod of any 2 edges from above


    // transform normal to worldspace
    //float3 hitPtNormal = normalize(make_float3((
    //    tri.transform.inverse()).transpose() * make_float4(N, 0)));
    float3 hitPtNormal = make_float3(tri.transform * make_float4(N, 0)); 

    // transform hit point to worldspace
    //float4 temp_hit = tri.transform * make_float4(hitPt, 1);
    //hitPt = make_float3(temp_hit / (float) temp_hit.w) /*+ epsilon * hitPtNormal*/;

    //// obtain parametric distance to hitPoint in worldspace
    t = length(hitPt - ray.origin);

    // compute reflection ray direction
    float3 reflectionDir = normalize(ray.direction - (2.0f * dot(ray.direction, hitPtNormal) * hitPtNormal));
    //float3 reflectionDir = normalize(ray.direction - 2.0f * dot(ray.direction, N) * N);

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // TODO: assign attribute variables here
        // Pass attributes: i.e. materials of the object
        attrib = tri.attributes;
        intersectData.hitPoint = hitPt;
        intersectData.hitPointNormal = /*hitPtNormal*/N;
        intersectData.reflectDir = reflectionDir;
        intersectData.rayDir = ray.direction;
        intersectData.rayOrig = ray.origin;
        // Pass hitPt and normal at hitPt into payload 
        // to calculate payload.radiance in closestHit()
        //payload.hitPoint = hitPt;
        //payload.hitPointNormal = hitPtNormal;
        //payload.dir = reflectionDir;

        //shadowPayload.isVisible = true;

        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Triangle tri = triangles[primIndex];

    result[0] = -1000.f;
    result[1] = -1000.f;
    result[2] = -1000.f;
    result[3] = 1000.f;
    result[4] = 1000.f;
    result[5] = 1000.f;

    // TODO: implement triangle bouding box
    
    // find the x, y, and z max and mins:
    float xMax, yMax, zMax, xMin, yMin, zMin;
    
    float tri0x = tri.vertices[0].x;
    float tri1x = tri.vertices[1].x;
    float tri2x = tri.vertices[2].x;

    float tri0y = tri.vertices[0].y;
    float tri1y = tri.vertices[1].y;
    float tri2y = tri.vertices[2].y;

    float tri0z = tri.vertices[0].z;
    float tri1z = tri.vertices[1].z;
    float tri2z = tri.vertices[2].z;

    // if tri0x > tri1x, we test if tri0x also greater than tri2x, if so, tri0x is greatest
    // else tri1x > tri0x, we test if tri1x also greater than tri2x, if so, tri1x greatest, 
    // else tri2x greatest. Rinse and repeat for all Min and Max
    xMax = (tri0x > tri1x) ? ((tri0x > tri2x) ? tri0x : tri2x) : ((tri1x > tri2x) ? tri1x : tri2x);
    yMax = (tri0y > tri1y) ? ((tri0y > tri2y) ? tri0y : tri2y) : ((tri1y > tri2y) ? tri1y : tri2y);
    zMax = (tri0z > tri1z) ? ((tri0z > tri2z) ? tri0z : tri2z) : ((tri1z > tri2z) ? tri1z : tri2z);

    xMin = (tri0x < tri1x) ? ((tri0x < tri2x) ? tri0x : tri2x) : ((tri1x < tri2x) ? tri1x : tri2x);
    yMin = (tri0y < tri1y) ? ((tri0y < tri2y) ? tri0y : tri2y) : ((tri1y < tri2y) ? tri1y : tri2y);
    zMin = (tri0z < tri1z) ? ((tri0z < tri2z) ? tri0z : tri2z) : ((tri1z < tri2z) ? tri1z : tri2z);

    result[0] = xMin;
    result[1] = yMin;
    result[2] = zMin;
    result[3] = xMax;
    result[4] = yMax;
    result[5] = zMax;
}
