#include "hip/hip_runtime.h"
#include "optix.h"
#include "optix_device.h"
#include "Geometries.h"
#include "Payloads.h"

using namespace optix;

rtBuffer<Triangle> triangles; // a buffer of all triangles 

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

// Transfer values into payload variable for color calc in closestHit()
rtDeclareVariable(Payload, payload, rtPayload, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and triangle
    Triangle tri = triangles[primIndex];
    float t;

    // TODO: implement triangle intersection test here
    
    // get plane normal
    float3 edge1 = tri.vertices[1] - tri.vertices[0]; 
    float3 edge2 = tri.vertices[2] - tri.vertices[0]; 
    // we have normal of triangle, N
    float3 N = normalize(cross(edge1, edge2)); 

    // triangles don't need to worry about transforming
    // ray by M-1 since it is still a triangle (but for spheres, 
    // must consider)
    float4 temp_ray = tri.transform.inverse() * make_float4(ray.origin, 1);
    float3 ray_orig = make_float3(temp_ray / (float)temp_ray.w);
    //ray_orig = make_float3(ray)
    float3 ray_dir = normalize(make_float3(tri.transform.inverse() * make_float4(ray.direction, 0)));

    // find parametric dist t: 
    //t = (dot(tri.vertices[0], N) - dot(ray.origin, N)) / dot(ray.direction, N);
    t = (dot(tri.vertices[0], N) - dot(ray_orig, N)) / dot(ray_dir, N);
    if (t < 0) {
        // triangle is not positive distance to ray, i.e. behind
        return; 
    }

    // check if ray is within the triangle, use barycentric
    // we use cross prod of an edge in the tri and the hit point
    // we know hit point is outside of triangle if dot product of 
    // Normal and orthogonal vect is < 0
    // Note: tri is ACW 
    float epsilon = 0.001f;
    float3 hitPt = ray_orig + t * ray_dir + N*epsilon; // account for shadow acne: 
    
    float3 orthogEdge;
    // check 1 (total 3 edges to check) 
    float3 edgeV1V0 = tri.vertices[1] - tri.vertices[0]; 
    float3 edgePV0 = hitPt - tri.vertices[0]; 
    orthogEdge = cross(edgeV1V0, edgePV0); 
    // calc u, v or w here: 

    // check if hitPt inside or outside tri 
    if (dot(N, orthogEdge) < 0)
        return; // hitPt outside

    // Check next edge
    float3 edgeV2V1 = tri.vertices[2] - tri.vertices[1];
    float3 edgePV1 = hitPt - tri.vertices[1];
    orthogEdge = cross(edgeV2V1, edgePV1);

    if (dot(N, orthogEdge) < 0)
        return; 

    // last check: edgeV0V2 and edgePV2
    float3 edgeV0V2 = tri.vertices[0] - tri.vertices[2]; 
    float3 edgePV2 = hitPt - tri.vertices[2];
    orthogEdge = cross(edgeV0V2, edgePV2);

    if (dot(N, orthogEdge) < 0)
        return;

    // made it here, means the hit point is inside the tri!

    // find normal at hitPoint, pass into payload
    // Note: the triangle already transformed during parsing
    //float3 hitPt = ray.origin + t * ray.direction; 
    // cross prod of any 2 edges from above

    // transform normal to worldspace
    float3 hitPtNormal = normalize(make_float3((
        tri.transform.inverse()).transpose() * make_float4(N, 0)));
    // transform hit point to worldspace
    float4 temp_hit = tri.transform * make_float4(hitPt, 1);
    hitPt = make_float3(temp_hit / (float) temp_hit.w);

    // compute reflection ray direction
    float3 reflectionDir = normalize(ray.direction - 2.0f * dot(ray.direction, hitPtNormal) * hitPtNormal);

    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // TODO: assign attribute variables here
        // Pass attributes: i.e. materials of the object
        attrib = tri.attributes;

        // Pass hitPt and normal at hitPt into payload 
        // to calculate payload.radiance in closestHit()
        payload.hitPoint = hitPt;
        payload.hitPointNormal = hitPtNormal;
        payload.dir = reflectionDir;

        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Triangle tri = triangles[primIndex];

    // TODO: implement triangle bouding box
    result[0] = -1000.f;
    result[1] = -1000.f;
    result[2] = -1000.f;
    result[3] = 1000.f;
    result[4] = 1000.f;
    result[5] = 1000.f;
}