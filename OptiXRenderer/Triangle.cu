#include "hip/hip_runtime.h"
#include "optix.h"
#include "optix_device.h"
#include "Geometries.h"
#include "Payloads.h"

using namespace optix;

rtBuffer<Triangle> triangles; // a buffer of all triangles 

rtDeclareVariable(Ray, ray, rtCurrentRay, );

// Attributes to be passed to material programs 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

// Transfer values into payload variable for color calc in closestHit()
rtDeclareVariable(Payload, payload, rtPayload, );

RT_PROGRAM void intersect(int primIndex)
{
    // Find the intersection of the current ray and triangle
    Triangle tri = triangles[primIndex];
    float t;

    // TODO: implement triangle intersection test here
    
    // get plane normal
    float3 edge1 = tri.vertices[1] - tri.vertices[0]; 
    float3 edge2 = tri.vertices[2] - tri.vertices[0]; 
    // we have normal of triangle, N
    float3 N = normalize(cross(edge1, edge2)); 

    // triangles don't need to worry about transforming
    // ray by M-1 since it is still a triangle (but for spheres, 
    // must consider)

    // find parametric dist t: 
    t = (dot(tri.vertices[0], N) - dot(ray.origin, N)) / dot(ray.direction, N);
    if (t < 0) {
        // triangle is not positive distance to ray, i.e. behind
        return; 
    }

    // check if ray is within the triangle, use barycentric
    // we use cross prod of an edge in the tri and the hit point
    // we know hit point is outside of triangle if dot product of 
    // Normal and orthogonal vect is < 0
    // Note: tri is ACW 
    float3 hitPt = ray.origin + t * ray.direction; 

    float3 orthogEdge;
    // check 1 (total 3 edges to check) 
    float3 edgeV1V0 = tri.vertices[1] - tri.vertices[0]; 
    float3 edgePV0 = hitPt - tri.vertices[0]; 
    orthogEdge = cross(edgeV1V0, edgePV0); 
    // calc u, v or w here: 

    // check if hitPt inside or outside tri 
    if (dot(N, orthogEdge) < 0)
        return; // hitPt outside

    // Check next edge
    float3 edgeV2V1 = tri.vertices[2] - tri.vertices[1];
    float3 edgePV1 = hitPt - tri.vertices[1];
    orthogEdge = cross(edgeV2V1, edgePV1);

    if (dot(N, orthogEdge) < 0)
        return; 

    // last check: edgeV0V2 and edgePV2
    float3 edgeV0V2 = tri.vertices[0] - tri.vertices[2]; 
    float3 edgePV2 = hitPt - tri.vertices[2];
    orthogEdge = cross(edgeV0V2, edgePV2);

    if (dot(N, orthogEdge) < 0)
        return;

    // made it here, means the hit point is inside the tri!

    // find normal at hitPoint, pass into payload
    // Note: the triangle already transformed during parsing
    //float3 hitPt = ray.origin + t * ray.direction; 
    // cross prod of any 2 edges from above
    float3 hitPtNormal = N;


    // Report intersection (material programs will handle the rest)
    if (rtPotentialIntersection(t))
    {
        // TODO: assign attribute variables here
        // Pass attributes: i.e. materials of the object
        attrib = tri.attributes;

        // Pass hitPt and normal at hitPt into payload 
        // to calculate payload.radiance in closestHit()
        payload.hitPoint = hitPt;
        payload.hitPointNormal = hitPtNormal;

        rtReportIntersection(0);
    }
}

RT_PROGRAM void bound(int primIndex, float result[6])
{
    Triangle tri = triangles[primIndex];

    // TODO: implement triangle bouding box
    result[0] = -1000.f;
    result[1] = -1000.f;
    result[2] = -1000.f;
    result[3] = 1000.f;
    result[4] = 1000.f;
    result[5] = 1000.f;
}