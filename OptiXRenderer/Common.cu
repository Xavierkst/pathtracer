#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>

#include "Payloads.h"

using namespace optix;

rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(float3, backgroundColor, , );

RT_PROGRAM void miss()
{
    // Set the result to be the background color if miss
    // TODO: change the color to backgroundColor
    //backgroundColor = make_float3(0, 0, 1);
    //payload.radiance = make_float3(1, 0, 0);
    payload.radiance = backgroundColor; 
    payload.done = true;
    //rtPrintf("miss!");
}

RT_PROGRAM void exception()
{
    // Print any exception for debugging
    const unsigned int code = rtGetExceptionCode();
    rtPrintExceptionDetails();
}

rtDeclareVariable(ShadowPayload, shadowPayload, rtPayload, );
rtDeclareVariable(float1, t, rtIntersectionDistance, );

RT_PROGRAM void anyHit()
{
    //rtPrintf("anyHit!");
    shadowPayload.isVisible = false;
    rtTerminateRay();
}