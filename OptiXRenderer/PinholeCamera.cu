#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Config.h"
#include "Light.h"

using namespace optix;

rtBuffer<float3, 2> resultBuffer; // used to store the render result
rtDeclareVariable(rtObject, root, , ); // Optix graph
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, ); // a 2d index (x, y)
rtDeclareVariable(int1, frameID, , );
rtBuffer<Config> config; // Config
rtDeclareVariable(int, samples_per_pixel, , );
rtDeclareVariable(uint, next_event_est, , );

RT_PROGRAM void generateRays()
{
    size_t2 resultSize = resultBuffer.size();
    unsigned int index = launchIndex.x * resultSize.y + launchIndex.y;
    unsigned int seed = tea<16>(index * frameID.x, 0);
    Config cf = config[0];
    float3 result = make_float3(0.f);
    float2 xy = make_float2(launchIndex);
    // xy.x += frameID.x == 1 ? 0.5f : rnd(seed);
    // xy.y += frameID.x == 1 ? 0.5f : rnd(seed);
    float2 ab;
    float3 dir;
    float3 origin;
    Payload payload;
    int i = 0;
    // cf.maxDepth = RT_DEFAULT_MAX;
    // Cast samples_per_pixel number of rays thru pixel xy
    for (int j = 0; j < samples_per_pixel; ++j) {
        // Prepare new payload for each sample
        payload.radiance = make_float3(.0f);
        payload.throughput = make_float3(1.0f);
        payload.depth = 0;
        payload.done = false;
        // Compute the ray direction: 
        xy = make_float2(launchIndex);
        // for th very first sample, we keep it at the center of the pixel
        // For every subsequent sample, jitter rays entering pixel
        xy.x += (j == 0) ? 0.5f : rnd(seed);
        xy.y += (j == 0) ? 0.5f : rnd(seed);
        ab = cf.tanHFov * (xy - cf.hSize) / cf.hSize; // calculates NDC coordinates -1 to +1 
        origin = cf.eye;
        dir = normalize(ab.x * cf.u + ab.y * cf.v - cf.w); // ray direction

        // For each pixel sample, we trace the path up to a depth D == cf.maxDepth
        do
        {
            payload.seed = tea<16>(index * frameID.x, i++);
            // Cast primary ray into the scene
            Ray ray = make_Ray(origin, dir, 0, cf.epsilon, RT_DEFAULT_MAX);
            rtTrace(root, ray, payload); // Goes to intersection program for Sphere and Tri
            // Accumulate radiance
            result += payload.radiance;
            payload.radiance = make_float3(0.f);
            // Continue "walking" thru the scene from one hit point to another
            origin = payload.origin; 
            dir = payload.dir;
        } while (!payload.done && payload.depth != cf.maxDepth);
    }
    
    // average out the results 
    result = (result / samples_per_pixel);
    result = make_float3(powf(result.x, 1.0f / cf.gamma), powf(result.y, 1.0f / cf.gamma), powf(result.z, 1.0f / cf.gamma));
    
    if (frameID.x == 1) 
        resultBuffer[launchIndex] = result;
    else
    {
        float u = 1.0f / (float)frameID.x;
        float3 oldResult = resultBuffer[launchIndex];
        resultBuffer[launchIndex] = lerp(oldResult, result, u);
    }
}