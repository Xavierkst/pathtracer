#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

#include "Payloads.h"
#include "Camera.h"

using namespace optix;

rtBuffer<float3, 2> resultBuffer; // used to store the render result

rtDeclareVariable(rtObject, root, , ); // Optix graph

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, ); // a 2d index (x, y)

rtDeclareVariable(int1, frameID, , );

// Camera info 

// TODO:: delcare camera variables here
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float, fovy, , );
rtDeclareVariable(int, width, , );
rtDeclareVariable(int, height, , );

//rtPrintf("%d", resultBuffer.size());

RT_PROGRAM void generateRays()
{
    size_t2 screen_size = resultBuffer.size();
    //rtPrintf("the total number of pixels is: %d", screen_size);

    float3 result = make_float3(0.f);

    // TODO: calculate the ray direction (change the following lines)
    float3 origin = eye;  // origin should be pos of camera
    
    //float2 d = make_float2(launchIndex) / make_float2(screen_size) * 2.0f - 1.0f;
    float aspectRatio = (float) width / (float)height;
    float alpha = ((2.0f * ((float)launchIndex.x + 0.5f) / (float)width) - 1.0f) * tan(fovy / 2.0f) * aspectRatio;
    float beta = (1.0f - (2.0f * ((float)launchIndex.y + 0.5f) / (float)height) ) * tan(fovy/2.0f);
    //rtPrintf("alpha beta: %f %f", alpha, beta);

    //float3 dir = make_float3(0, 0, 1); // dir should be toward some (i,j)-th cell?
    float3 dir = normalize(alpha * U + beta * V - W);

    float epsilon = 0.001f; 

    // TODO: modify the following lines if you need
    // Shoot a ray to compute the color of the current pixel
    Ray ray = make_Ray(origin, dir, 0, epsilon, RT_DEFAULT_MAX);
    Payload payload;
    rtTrace(root, ray, payload);

    // Write the result
    resultBuffer[launchIndex] = payload.radiance;
}