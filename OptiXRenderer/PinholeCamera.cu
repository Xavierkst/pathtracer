#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

#include "Payloads.h"
#include "Camera.h"

using namespace optix;

rtBuffer<float3, 2> resultBuffer; // used to store the render result

rtDeclareVariable(rtObject, root, , ); // Optix graph

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, ); // a 2d index (x, y)

rtDeclareVariable(int1, frameID, , );

// Camera info 

// TODO:: delcare camera variables here
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float, fovy, , );
rtDeclareVariable(int, width, , );
rtDeclareVariable(int, height, , );
rtDeclareVariable(int, depth, , );

//rtPrintf("%d", resultBuffer.size());

//rtDeclareVariable(intersectionData, intersectData, attribute intersectData, );

RT_PROGRAM void generateRays()
{
    float3 result = make_float3(0.f);
    // TODO: calculate the ray direction (change the following lines)
    float3 origin = eye;  // origin should be pos of camera
    float aspectRatio = (float) width / (float)height;
    float alpha = ((2.0f * ((float)launchIndex.x + 0.5f) / (float)width) - 1.0f) * tan(fovy / 2.0f) * aspectRatio;
    float beta = ((2.0f * ((float)launchIndex.y + 0.5f) / (float)height) - 1.0f) * tan(fovy / 2.0f);

    float3 dir = normalize(alpha * U + beta * V - W);

    float epsilon = 0.001f; 
    Payload payload;
    payload.done = false;
    payload.depth = depth;
    payload.spec = make_float3(.0f);
    // TODO: modify the following lines if you need
    // Shoot a ray to compute the color of the current pixel
    //Ray ray = make_Ray(origin, dir, 0, epsilon, RT_DEFAULT_MAX);
    //rtTrace(root, ray, payload);
    //result = payload.radiance;
    //Ray ray = make_Ray(origin, dir, 0, epsilon, RT_DEFAULT_MAX);
    //rtTrace(root, ray, payload);

    do {
        //rtPrintf("payload depth: %d and depth: %d\n", payload.depth, depth);
        Ray ray2 = make_Ray(origin, dir, 0, epsilon, RT_DEFAULT_MAX);
        rtTrace(root, ray2, payload);
        //result += make_float3(result.x * payload.radiance.x, result.y * payload.radiance.y, result.z * payload.radiance.z);
        result += payload.radiance;
         //set up for next ray cast
        origin = payload.rayOrigin; 
        dir = payload.rayDir;
        --payload.depth;
        //rtPrintf("%f, %f, %f and %f %f %f \n", payload.rayOrigin.x, payload.rayOrigin.y, payload.rayOrigin.z, temp_origin.x, temp_origin.y, temp_origin.z);
     } while (!payload.done && (payload.depth > 0));

    // Write the result
    resultBuffer[launchIndex] = result;
    //resultBuffer[launchIndex] = make_float3(.0f, .0f, .0f);
}