#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

#include "Payloads.h"

using namespace optix;

rtBuffer<float3, 2> resultBuffer; // used to store the render result

rtDeclareVariable(rtObject, root, , ); // Optix graph

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, ); // a 2d index (x, y)

rtDeclareVariable(int1, frameID, , );

// Camera info 

// TODO:: delcare camera variables here

RT_PROGRAM void generateRays()
{
    float3 result = make_float3(0.f);

    // TODO: calculate the ray direction (change the following lines)
    float3 origin = make_float3(0, 0, 0);  // origin should be pos of camera
    float3 dir = make_float3(0, 0, 1); // dir should be toward some (i,j)-th cell?
    float epsilon = 0.001f; 

    // TODO: modify the following lines if you need
    // Shoot a ray to compute the color of the current pixel
    Ray ray = make_Ray(origin, dir, 0, epsilon, RT_DEFAULT_MAX);
    Payload payload;
    rtTrace(root, ray, payload);

    // Write the result
    resultBuffer[launchIndex] = payload.radiance;
}