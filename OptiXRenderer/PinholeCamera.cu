#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>

#include "Payloads.h"
#include "Camera.h"

using namespace optix;

rtBuffer<float3, 2> resultBuffer; // used to store the render result

rtDeclareVariable(rtObject, root, , ); // Optix graph

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, ); // a 2d index (x, y)

rtDeclareVariable(int1, frameID, , );

// Camera info 

// TODO:: delcare camera variables here
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float, fovy, , );
rtDeclareVariable(int, width, , );
rtDeclareVariable(int, height, , );
rtDeclareVariable(int, depth, , );

//rtPrintf("%d", resultBuffer.size());

RT_PROGRAM void generateRays()
{
    float3 result = make_float3(0.f);
    // TODO: calculate the ray direction (change the following lines)
    float3 origin = eye;  // origin should be pos of camera
    float aspectRatio = (float) width / (float)height;
    float alpha = ((2.0f * ((float)launchIndex.x + 0.5f) / (float)width) - 1.0f) * tan(fovy / 2.0f) * aspectRatio;
    float beta = ((2.0f * ((float)launchIndex.y + 0.5f) / (float)height) - 1.0f) * tan(fovy / 2.0f);

    float3 dir = normalize(alpha * U + beta * V - W);

    float epsilon = 0.001f; 
    Payload payload;
    payload.depth = depth;
    // TODO: modify the following lines if you need
    // Shoot a ray to compute the color of the current pixel
    //Ray ray = make_Ray(origin, dir, 0, epsilon, RT_DEFAULT_MAX);
    //rtTrace(root, ray, payload);
    //result = payload.radiance;


    do {
        Ray ray = make_Ray(origin, dir, 0, epsilon, RT_DEFAULT_MAX);
        rtTrace(root, ray, payload);
        result += payload.radiance;
        // set up for next ray cast
        origin = payload.hitPoint; 
        dir = payload.dir;
        //--payload.depth;
    } while (!payload.done && payload.depth > 0);

    // Write the result
    resultBuffer[launchIndex] = result;
}