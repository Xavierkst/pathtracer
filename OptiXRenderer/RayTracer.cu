#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"
#include "Config.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;
rtBuffer<QuadLight> qlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

rtBuffer<Config> config; // Config

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

rtDeclareVariable(uint, light_samples, , );
rtDeclareVariable(uint, light_stratify, , );

RT_PROGRAM void closestHit()
{
    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float3 result = mv.ambient + mv.emission;
    float3 sample_result = make_float3(.0f);

    // Calculate the direct illumination of point lights
    for (int i = 0; i < plights.size(); i++)
    {
        // Shoot a shadow to determin whether the object is in shadow
        float3 lightDir = normalize(plights[i].location - attrib.intersection);
        float lightDist = length(plights[i].location - attrib.intersection);
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;
        Ray shadowRay = make_Ray(attrib.intersection + lightDir * cf.epsilon, 
            lightDir, 1, cf.epsilon, lightDist);
        rtTrace(root, shadowRay, shadowPayload);

        // If not in shadow
        if (shadowPayload.isVisible)
        {
            float3 H = normalize(lightDir + attrib.wo);
            float att = dot(plights[i].attenuation, make_float3(1, lightDist, lightDist * lightDist));
            float3 I = mv.diffuse * fmaxf(dot(attrib.normal, lightDir), 0);
            I += mv.specular * pow(fmaxf(dot(attrib.normal, H), 0), mv.shininess);
            I *= plights[i].color / att;
            result += I;
        }
    }

    // Calculate the direct illumination of directional lights
    for (int i = 0; i < dlights.size(); i++)
    {
        // Shoot a shadow to determin whether the object is in shadow
        float3 lightDir = dlights[i].direction;
        float lightDist = RT_DEFAULT_MAX;
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;
        Ray shadowRay = make_Ray(attrib.intersection + lightDir * cf.epsilon, 
            lightDir, 1, cf.epsilon, lightDist);
        rtTrace(root, shadowRay, shadowPayload);

        // If not in shadow
        if (shadowPayload.isVisible)
        {
            float3 H = normalize(lightDir + attrib.wo);
            float3 I = mv.diffuse * fmaxf(dot(attrib.normal, lightDir), 0);
            I += mv.specular * pow(fmaxf(dot(attrib.normal, H), 0), mv.shininess);
            I *= dlights[i].color;
            result += I;
        }
    }

    for (int k = 0; k < qlights.size(); ++k) {
        // Compute direct lighting equation for w_i_k ray, for k = 1 to N*N
        
        // check if stratify or random sampling
        if (light_stratify) {
            // double for loop here 
            for (int i = 0; i < light_samples; ++i) {
                for (int j = 0; j < light_samples; ++j) {
                    float3 a = qlights[k].tri1.v1;
                    float3 b = qlights[k].tri1.v2;
                    float3 c = qlights[k].tri2.v2;
                    float3 d = qlights[k].tri1.v3;
                    
                    float3 ac = c - a;
                    float3 ab = b - a;
                    // generate random float vals u1 and u2
                    float u1 = rnd(payload.seed);
                    float u2 = rnd(payload.seed);

                    float3 ray_origin = attrib.intersection; 
                    float3 ray_dir = a + ((j + u1) * (ab / (float) light_samples)) + ((i + u2) * (ac / (float) light_samples));
                    Ray sample_ray = make_Ray(ray_origin, ray_dir, 0, cf.epsilon,  RT_DEFAULT_MAX);
                    rtTrace(root, sample_ray, payload);

                    // rendering equation here: 
                    //float 

                    //sample_result += payload. 
                }
            }
        }
        else {

        }

    }

    // Another for-loop here to calculate contribution of quadLights
    // eg. for (int i = 0; i < qlights.size(); ++i) {}
    //for (int i = 0; i < qlights.size(); ++i) {
    //    float3 f_brdf = mv.diffuse / M_PIf;// brdf function 
    //    float3 hitPt = attrib.intersection;
    //    float3 hitPtNormal = attrib.normal;

    //    float3 a = qlights[i].tri1.v1;
    //    float3 b = qlights[i].tri1.v2;
    //    float3 c = qlights[i].tri2.v2;
    //    float3 d = qlights[i].tri1.v3;
    //    
    //    float3 points[] = { qlights[i].tri1.v1, qlights[i].tri1.v2, qlights[i].tri2.v2, qlights[i].tri1.v3 };

    //    float3 p1 = points[0]; float3 p2 = points[1]; float3 p3 = points[2]; float3 p4 = points[3];
    //    float theta_1 = acosf(dot(normalize(p1 - hitPt), normalize(p2 - hitPt)));
    //    float theta_2 = acosf(dot(normalize(p2 - hitPt), normalize(p3 - hitPt)));
    //    float theta_3 = acosf(dot(normalize(p3 - hitPt), normalize(p4 - hitPt)));
    //    float theta_4 = acosf(dot(normalize(p4 - hitPt), normalize(p1 - hitPt)));

    //    float3 gamma_1 = normalize(cross((p1 - hitPt), (p2 - hitPt)));
    //    float3 gamma_2 = normalize(cross((p2 - hitPt), (p3 - hitPt)));
    //    float3 gamma_3 = normalize(cross((p3 - hitPt), (p4 - hitPt)));
    //    float3 gamma_4 = normalize(cross((p4 - hitPt), (p1 - hitPt)));

    //    float3 irradiance_vec = 0.5f * ((theta_1 * gamma_1) + 
    //        (theta_2 * gamma_2) + (theta_3 * gamma_3) + (theta_4 * gamma_4));

    //    float3 dir_radiance = f_brdf * qlights[i].color * dot(irradiance_vec, hitPtNormal);
    //    result += dir_radiance;

    //}

    // Compute the final radiance
    payload.radiance = result * payload.throughput;

    // Calculate reflection
    if (length(mv.specular) > 0)
    {
        // Set origin and dir for tracing the reflection ray
        payload.origin = attrib.intersection;
        payload.dir = reflect(-attrib.wo, attrib.normal); // mirror reflection

        payload.depth++;
        payload.throughput *= mv.specular;
    }
    else
    {
        payload.done = true;
    }
}