#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );
rtDeclareVariable(intersectionData, intersectData, attribute intersectData, );
rtDeclareVariable(float1, t, rtIntersectionDistance, );
rtDeclareVariable(int, depth, , ); // depth from Renderer.cpp

RT_PROGRAM void closestHit()
{
    // TODO: calculate the color using the Blinn-Phong reflection model
    float epsilon = .001f;
    float3 result = attrib.ambient + attrib.emission;
     
    for (int i = 0; i < plights.size(); i++) {
        //cast shadow ray
        float3 lightVec = (plights[i].light_pos - intersectData.hitPoint);
        float distToLight = length(lightVec);
        lightVec = normalize(lightVec);

        // create shadow ray and cast it
        float3 shadowRayOrigin = intersectData.hitPoint + intersectData.hitPointNormal * epsilon;
    	Ray shadowRay = make_Ray(shadowRayOrigin, lightVec, 1, epsilon, distToLight);
        ShadowPayload shadowPayload; 
        shadowPayload.isVisible = true;
    	rtTrace(root, shadowRay, shadowPayload); 

        // half angle = normalize of L + V, L (hitPt to light), 
        // V (view dir, eye / prev hitPt to hitPt)
        float3 half_angle = normalize(lightVec + normalize(intersectData.rayOrig - intersectData.hitPoint));

        // compute blinn-phong
        if (shadowPayload.isVisible) {
            //result = (half_angle);
            result += (plights[i].light_color /
                (plights[i].attenuation.constant + plights[i].attenuation.linear * distToLight +
                    plights[i].attenuation.quadratic * powf(distToLight, 2.0f))) * 
                (attrib.diffuse * fmaxf(dot(intersectData.hitPointNormal,
                        normalize(lightVec)), .0f) +
                        attrib.specular * powf(fmaxf(dot( intersectData.hitPointNormal, half_angle), .0f),
                            attrib.shininess));
        }
    }

    for (int i = 0; i < dlights.size(); i++) {
        //cast shadow ray
    	float distToLight = RT_DEFAULT_MAX;
        float3 shadowRayOrigin = intersectData.hitPoint + intersectData.hitPointNormal * epsilon;
    	Ray shadowRay = make_Ray(shadowRayOrigin, normalize(dlights[i].light_dir), 1, epsilon, distToLight);
    	//Ray shadowRay = make_Ray(intersectData.hitPoint, -dlights[i].light_dir, 1, epsilon, distToLight);
        ShadowPayload shadowPayload; 
        shadowPayload.isVisible = true;
    	rtTrace(root, shadowRay, shadowPayload); 

        // half angle = normalize of L + V, L (hitPt to light), 
        float3 half_angle = normalize(normalize(dlights[i].light_dir) + (intersectData.rayOrig - intersectData.hitPoint));

        // compute blinn-phong
        if (shadowPayload.isVisible) {
            result += (dlights[i].light_color * (attrib.diffuse * fmaxf(
                        dot(intersectData.hitPointNormal, normalize(dlights[i].light_dir)), .0f) +
                        attrib.specular * powf(fmaxf(dot(intersectData.hitPointNormal, half_angle), .0f),
                            attrib.shininess)));
        }
    }
    // pass the new ray dir and reflection dir into payload 
    // to be used in rayGeneration do-While loop: 
    payload.rayOrigin = intersectData.hitPoint + intersectData.hitPointNormal * epsilon;
    payload.rayDir = intersectData.reflectDir;

    if (payload.depth == depth) {
        payload.radiance = result;
        payload.spec = attrib.specular;
    }
    else {
        payload.radiance = payload.spec * result;
        // accumulate specular: 
        // r_1 + S_1 * r2 + S_1 * S_2 * r3 + ...
        payload.spec *= attrib.specular;
    }
}