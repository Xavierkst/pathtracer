#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );
rtDeclareVariable(intersectionData, intersectData, attribute intersectData, );
//rtDeclareVariable(ShadowPayload, shadowPayload, rtPayload, );
rtDeclareVariable(float1, t, rtIntersectionDistance, );
//rtDeclareVariable(float3, eye, , );
// depth from Renderer.cpp
rtDeclareVariable(int, depth, , );

RT_PROGRAM void closestHit()
{
    //rtPrintf("radiance: %f %f %f\n", payload.radiance.x, payload.radiance.y, payload.radiance.z);
    // TODO: calculate the color using the Blinn-Phong reflection model
    float epsilon = .001f;
    float3 result = attrib.ambient + attrib.emission;
    //printf("%f, %f, %f\n", attrib.ambient.x, attrib.ambient.y, attrib.ambient.z);
     
    // QUESTION: is the HALF-ANGLE: H = normalize(L + V), where L is the direction from hitpoint to light, and V is dir from hitPt to eye?
    for (int i = 0; i < plights.size(); i++) {
        //cast shadow ray
        float3 lightVec = (plights[i].light_pos - intersectData.hitPoint);
        float distToLight = length(lightVec);
        lightVec = normalize(lightVec);

        // create shadow ray and cast it
        float3 shadowRayOrigin = intersectData.hitPoint + intersectData.hitPointNormal * epsilon;
    	Ray shadowRay = make_Ray(shadowRayOrigin, lightVec, 1, epsilon, distToLight);
        ShadowPayload shadowPayload; 
        shadowPayload.isVisible = true;
    	rtTrace(root, shadowRay, shadowPayload); 

        float3 half_angle = normalize(lightVec + normalize(intersectData.rayOrig - intersectData.hitPoint));
        //float3 half_angle = normalize((-intersectData.rayDir) + (eye - intersectData.hitPoint));

        if (shadowPayload.isVisible) {
            //result = (half_angle);
            result += (plights[i].light_color /
                (plights[i].attenuation.constant + plights[i].attenuation.linear * distToLight +
                    plights[i].attenuation.quadratic * powf(distToLight, 2.0f))) * 
                (attrib.diffuse * fmaxf(dot(intersectData.hitPointNormal,
                        normalize(lightVec)), .0f) +
                        attrib.specular * powf(fmaxf(dot( intersectData.hitPointNormal, half_angle), .0f),
                            attrib.shininess));
        }
    }

    for (int i = 0; i < dlights.size(); i++) {
        //cast shadow ray
    	float distToLight = RT_DEFAULT_MAX;
        float3 shadowRayOrigin = intersectData.hitPoint + intersectData.hitPointNormal * epsilon;
    	Ray shadowRay = make_Ray(shadowRayOrigin, normalize(dlights[i].light_dir), 1, epsilon, distToLight);
    	//Ray shadowRay = make_Ray(intersectData.hitPoint, -dlights[i].light_dir, 1, epsilon, distToLight);
        ShadowPayload shadowPayload; 
        shadowPayload.isVisible = true;
    	rtTrace(root, shadowRay, shadowPayload); 

        float3 half_angle = normalize(normalize(dlights[i].light_dir) + (intersectData.rayOrig - intersectData.hitPoint));
        //float3 half_angle = normalize(-intersectData.rayDir + (eye - intersectData.hitPoint));
        if (shadowPayload.isVisible) {
            result += (dlights[i].light_color * (attrib.diffuse * fmaxf(
                        dot(intersectData.hitPointNormal, normalize(dlights[i].light_dir)), .0f) +
                        attrib.specular * powf(fmaxf(dot(intersectData.hitPointNormal, half_angle), .0f),
                            attrib.shininess)));
        }
    }
    // pass the new ray dir and reflection dir into payload 
    // to be used in rayGeneration do-While loop: 
    payload.rayOrigin = intersectData.hitPoint /*+ epsilon * intersectData.hitPointNormal*/;
    payload.rayDir = intersectData.reflectDir;

    //rtPrintf("payload depth: %d and depth: %d\n", payload.depth, depth);
    if (payload.depth == depth) {
        payload.radiance = result;
        payload.spec = attrib.specular;
    }
    else {
        //rtPrintf("spec: %f %f %f\n", payload.spec.x, payload.spec.y, payload.spec.z);
        payload.radiance = payload.spec * result;
        payload.spec *= attrib.specular;
    }
    //rtPrintf("radiance: %f %f %f\n", attrib.specular.x, attrib.specular.y, attrib.specular.z);
    //--payload.depth;
}