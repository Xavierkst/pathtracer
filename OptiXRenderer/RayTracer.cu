#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"
#include "Config.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;
rtBuffer<QuadLight> qlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

rtBuffer<Config> config; // Config

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

rtDeclareVariable(uint, light_samples, , );
rtDeclareVariable(uint, light_stratify, , );

RT_PROGRAM void closestHit()
{
    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float3 result = mv.ambient + mv.emission;

    // Calculate the direct illumination of point lights
    for (int i = 0; i < plights.size(); i++)
    {
        // Shoot a shadow to determin whether the object is in shadow
        float3 lightDir = normalize(plights[i].location - attrib.intersection);
        float lightDist = length(plights[i].location - attrib.intersection);
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;
        Ray shadowRay = make_Ray(attrib.intersection + lightDir * cf.epsilon, 
            lightDir, 1, cf.epsilon, lightDist);
        rtTrace(root, shadowRay, shadowPayload);

        // If not in shadow
        if (shadowPayload.isVisible)
        {
            float3 H = normalize(lightDir + attrib.wo);
            float att = dot(plights[i].attenuation, make_float3(1, lightDist, lightDist * lightDist));
            float3 I = mv.diffuse * fmaxf(dot(attrib.normal, lightDir), 0);
            I += mv.specular * pow(fmaxf(dot(attrib.normal, H), 0), mv.shininess);
            I *= plights[i].color / att;
            result += I;
        }
    }

    // Calculate the direct illumination of directional lights
    for (int i = 0; i < dlights.size(); i++)
    {
        // Shoot a shadow to determin whether the object is in shadow
        float3 lightDir = dlights[i].direction;
        float lightDist = RT_DEFAULT_MAX;
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;
        Ray shadowRay = make_Ray(attrib.intersection + lightDir * cf.epsilon, 
            lightDir, 1, cf.epsilon, lightDist);
        rtTrace(root, shadowRay, shadowPayload);

        // If not in shadow
        if (shadowPayload.isVisible)
        {
            float3 H = normalize(lightDir + attrib.wo);
            float3 I = mv.diffuse * fmaxf(dot(attrib.normal, lightDir), 0);
            I += mv.specular * pow(fmaxf(dot(attrib.normal, H), 0), mv.shininess);
            I *= dlights[i].color;
            result += I;
        }
    }

    // Compute the final radiance
    payload.radiance = result * payload.throughput;

    // Calculate reflection
    if (length(mv.specular) > 0)
    {
        // Set origin and dir for tracing the reflection ray
        payload.origin = attrib.intersection;
        payload.dir = reflect(-attrib.wo, attrib.normal); // mirror reflection

        payload.depth++;
        payload.throughput *= mv.specular;
    }
    else
    {
        payload.done = true;
    }
    //payload.done = true;
}

RT_PROGRAM void analyticDirect() {

    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float3 result = mv.ambient + mv.emission;

    // for-loop here to calculate contribution of quadLights
    // if no light samples, do analytical direct:
    if (light_samples == 0) {
        for (int i = 0; i < qlights.size(); ++i) {
            float3 f_brdf = mv.diffuse / M_PIf;// brdf function 
            float3 hitPt = attrib.intersection;
            float3 hitPtNormal = attrib.normal;

            float3 a = qlights[i].tri1.v1;
            float3 b = qlights[i].tri1.v2;
            float3 c = qlights[i].tri2.v2;
            float3 d = qlights[i].tri1.v3;

            float3 points[] = { qlights[i].tri1.v1, qlights[i].tri1.v2, qlights[i].tri2.v2, qlights[i].tri1.v3 };

            float3 p1 = points[0]; float3 p2 = points[1]; float3 p3 = points[2]; float3 p4 = points[3];
            float theta_1 = acosf(dot(normalize(p1 - hitPt), normalize(p2 - hitPt)));
            float theta_2 = acosf(dot(normalize(p2 - hitPt), normalize(p3 - hitPt)));
            float theta_3 = acosf(dot(normalize(p3 - hitPt), normalize(p4 - hitPt)));
            float theta_4 = acosf(dot(normalize(p4 - hitPt), normalize(p1 - hitPt)));

            float3 gamma_1 = normalize(cross((p1 - hitPt), (p2 - hitPt)));
            float3 gamma_2 = normalize(cross((p2 - hitPt), (p3 - hitPt)));
            float3 gamma_3 = normalize(cross((p3 - hitPt), (p4 - hitPt)));
            float3 gamma_4 = normalize(cross((p4 - hitPt), (p1 - hitPt)));

            float3 irradiance_vec = 0.5f * ((theta_1 * gamma_1) +
                (theta_2 * gamma_2) + (theta_3 * gamma_3) + (theta_4 * gamma_4));

            float3 dir_radiance = f_brdf * qlights[i].color * dot(irradiance_vec, hitPtNormal);
            result += dir_radiance;
        }
    }

    payload.radiance = result;

    payload.done = true;
}


RT_PROGRAM void direct() {

    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float3 result = mv.ambient + mv.emission;

    for (int k = 0; k < qlights.size(); ++k) {
        float3 sampled_result = make_float3(0);
        // Compute direct lighting equation for w_i_k ray, for k = 1 to N*N
        float3 a = qlights[k].tri1.v1;
        float3 b = qlights[k].tri1.v2;
        float3 c = qlights[k].tri2.v3;
        float3 d = qlights[k].tri2.v2;

        float3 ac = c - a;
        float3 ab = b - a;
        float area = length(cross(ab, ac));
        int root_light_samples = (int)sqrtf(light_samples);
        // check if stratify or random sampling
        // double for loop here 
        for (int i = 0; i < root_light_samples; ++i) {
            for (int j = 0; j < root_light_samples; ++j) {
                // generate random float vals u1 and u2
                float u1 = rnd(payload.seed);
                float u2 = rnd(payload.seed);

                float3 sampled_light_pos;
                if (light_stratify) {
                    sampled_light_pos = a + ((j + u1) * (ab / (float)root_light_samples)) +
                        ((i + u2) * (ac / (float)root_light_samples));
                }
                else {
                    sampled_light_pos = a + u1 * ab + u2 * ac;
                }
                float3 shadow_ray_origin = attrib.intersection /*+ attrib.normal * cf.epsilon*/;
                float3 shadow_ray_dir = normalize(sampled_light_pos - shadow_ray_origin);
                float light_dist = length(sampled_light_pos - shadow_ray_origin);
                Ray shadow_ray = make_Ray(shadow_ray_origin, shadow_ray_dir, 1, cf.epsilon, light_dist - cf.epsilon);

                ShadowPayload shadow_payload;
                shadow_payload.isVisible = true;
                rtTrace(root, shadow_ray, shadow_payload);

                if (shadow_payload.isVisible) {
                    // rendering equation here: 
                    //float3 w_i = sampled_light_pos;
                    float3 f_brdf = (mv.diffuse / M_PIf) +
                        (mv.specular * ((mv.shininess + 2.0f) / (2.0f * M_PIf)) *
                            powf(fmaxf(dot(normalize(reflect(-attrib.wo, attrib.normal)), normalize(sampled_light_pos - shadow_ray_origin)), .0f), mv.shininess));

                    float3 x_prime = sampled_light_pos;
                    float3 x = shadow_ray_origin;
                    float3 n = attrib.normal;
                    //float3 n_light = normalize(qlights[k].tri1.normal);
                    float3 n_light = normalize(cross(ab, ac));
                    //n_light = dot(n_light, normalize(x_prime - x)) > .0f ? n_light : -n_light;

                    float R = length(x - x_prime);

                    // note: normal should point AWAY from the hitpoint, i.e. dot(n_light, x - x_prime) < 0
                    float G = (1.0f / powf(R, 2.0f)) * fmaxf(dot(n, normalize(x_prime - x)), .0f) *
                        (fmaxf(dot(n_light, normalize(x_prime - x)), .0f));

                    sampled_result += f_brdf * G;
                }
            }
        }
        result += qlights[k].color * sampled_result * (area / (float)light_samples);
    }
    //rtPrintf("throughput val: %f \n", payload.throughput);
    payload.radiance = result;

    payload.done = true;
}

RT_PROGRAM void pathtracer() {

    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float pdf = 1.0f;
    float3 brdf = make_float3(0);
    float3 result = make_float3(0);
    float3 L_d = make_float3(0);
    float3 L_e = mv.emission;
    float3 r = normalize(reflect(-attrib.wo, attrib.normal));
    float s_mean = (mv.specular.x + mv.specular.y + mv.specular.z)/3.0f;
    float d_mean = (mv.diffuse.x + mv.diffuse.y + mv.diffuse.z)/3.0f;
    float u0 = rnd(payload.seed);
    float u1 = rnd(payload.seed);
    float u2 = rnd(payload.seed);
    float t = 0.0f;

    if (s_mean + d_mean == 0) {
	if (mv.brdf == 1) t = 1.0f;
    }
    else {
	t = (mv.brdf == 0) ? s_mean/(s_mean+d_mean) : fmaxf(0.25f, s_mean/(s_mean+d_mean));
    }

    float theta;
    float phi;

    switch (cf.IS) {
        case 0: 
            phi = 2.0f * M_PIf * u2;
            theta = acosf(u1);
            break;
        case 1: 
            phi = 2.0f * M_PIf * u2;
            theta = acosf(sqrtf(u1));
            break;
        case 2: 
            phi = 2.0f * M_PIf * u2;
            if (u0 > t) 
                theta = acosf(sqrtf(u1)); // theta_diffuse
            else 
                theta = acosf(powf(u1, (1.0f / (mv.shininess + 1.0f)))); // theta_specular
            break;
    }

    float3 sampleVec = make_float3(cosf(phi)*sinf(theta), sinf(phi)*sinf(theta), cosf(theta));

    float3 n = normalize(attrib.normal);
    float3 w = ((u0 <= t) && cf.IS == 2 && mv.brdf == 0) ? r : n;
    float3 a = make_float3(0,1,0);
    a = fabsf(dot(a,w)) > .9f ? make_float3(1,0,0) : a;

    float3 u = normalize(cross(a, w));
    float3 v = cross(w,u);
    if (mv.brdf == 1) {
	float theta_h_sample = atanf((mv.roughness * sqrtf(u1)) / sqrtf(1.0f - u1));
	float phi_h_sample = 2.0f * M_PIf * u2;
	sampleVec = make_float3(cosf(phi_h_sample) * sinf(theta_h_sample), 
	       sinf(phi_h_sample)*sinf(theta_h_sample), 
	       cosf(theta_h_sample));
    }

    float3 wi = make_float3(1.0f); 

    // get randomized new ray dir -- choose a sampling method
    wi = (sampleVec.x * u + sampleVec.y * v + sampleVec.z * w);   

    if (mv.brdf == 1) {
        wi = (reflect(-attrib.wo, wi));
    }

    float3 bruh = make_float3(.0f);

    switch (cf.IS) {
        case 0: 
            brdf = (mv.diffuse / M_PIf) +
                (mv.specular * ((mv.shininess + 2.0f) / (2.0f * M_PIf)) *
                    powf(fmaxf(dot(r, wi), .0f), mv.shininess));
            pdf = 1.0f / (2.0f * M_PIf);
            bruh = (brdf * fmaxf(dot(n, wi), .0f) * (1.0f / pdf));
            break;

        case 1: 
            brdf = (mv.diffuse / M_PIf) +
                (mv.specular * ((mv.shininess + 2.0f) / (2.0f * M_PIf)) *
                    powf(fmaxf(dot(r, wi), .0f), mv.shininess));
            pdf = dot(n, wi) / (M_PIf);
            bruh = (brdf * fmaxf(dot(n, wi), .0f) * (1.0f / pdf));
            break;

        case 2: 
            // check the material whether to use mod-phong or GGX brdf
            if (mv.brdf == 0) {
                brdf = (mv.diffuse / M_PIf) +
		       (mv.specular * ((mv.shininess + 2.0f) / (2.0f * M_PIf)) *
                        powf(fmaxf(dot(r, wi), .0f), mv.shininess));

                pdf = ((1.0f - t) * (fmaxf(dot(n, wi), .0f) / M_PIf)) + 
			t * ((mv.shininess + 1.0f) / (2.0f * M_PIf)) * 
                        powf(fmaxf(dot(r, wi), .0f), mv.shininess);
            }
            else {
                // construct GGX BRDF: 
		float wi_dot_n = dot(wi, n);
		float wo_dot_n = dot(attrib.wo, n);
                if (wi_dot_n > .0f && wo_dot_n > .0f) {
                    float alpha = mv.roughness;
                    float3 h = normalize(wi + attrib.wo); // half angle: 
                    float theta_h = acosf(clamp(dot(h, n),0.0f,1.0f)); // not sure if need to clamp 0
                    // microfacet distribution function, D: 
                    float D = (alpha * alpha) / (M_PIf * powf(cosf(theta_h), 4.0f) *
			       powf((alpha * alpha) + powf(tanf(theta_h), 2.0f), 2.0f));
                    // shadow-masking function, G:  
                    float G_1_wi = (wi_dot_n > .0f) ? 2.0f / (1.0f + sqrtf(1.0f + (alpha * alpha) *
				    powf(tanf(acosf(clamp(dot(wi, n),0.0f,1.0f))), 2.0f))) : .0f;
                    float G_1_wo = (wo_dot_n > .0f) ? 2.0f / (1.0f + sqrtf(1.0f + (alpha * alpha) *
				    powf(tanf(acosf(clamp(dot(attrib.wo, n),0.0f,1.0f))), 2.0f))) : .0f;
                    float G = G_1_wi * G_1_wo;

                    // fresnel function, F:
                    float3 F = mv.specular + (make_float3(1.0f) - mv.specular) * powf(1.0f - dot(wi, h), 5.0f);
                    float3 f_brdf_GGX = (F * G * D) / (4.0f * wi_dot_n * wo_dot_n);
                    brdf = (mv.diffuse / M_PIf) + f_brdf_GGX;

                    pdf = fmaxf(((1 - t) * (wi_dot_n / M_PIf)) + ((t * D * dot(n, h)) / (4.0f * fmaxf(dot(h, wi),0.0f))),0.0f);
                }
                else brdf = make_float3(0.0f); // assume f zero otherwise
            }
            bruh = (brdf * fmaxf(dot(n,wi),0) * (1.0f / pdf));
            break;
    }


    for (int k = 0; k < (cf.NEE ? qlights.size()*cf.NEE : 0); ++k) {
        float3 sampled_result = make_float3(.0f);
	float3 brdf_cum = make_float3(0);
        // Compute direct lighting equation for w_i_k/2 ray, for k = 1 to N*N
	int whichLight = (k >= qlights.size()) ? k%qlights.size() : k;
        float3 a = qlights[whichLight].tri1.v1;
        float3 b = qlights[whichLight].tri1.v2;
        float3 c = qlights[whichLight].tri2.v3;
        float3 d = qlights[whichLight].tri2.v2;

        float3 ac = c - a + cf.epsilon;
        float3 ab = b - a + cf.epsilon;
        float area = length(cross(ab, ac));
        int root_light_samples = (int)sqrtf(light_samples);
        // check if stratify or random sampling
        // double for loop here 
        for (int i = 0; i < root_light_samples; ++i) {
            for (int j = 0; j < root_light_samples; ++j) {

                float3 sampled_light_pos = make_float3(0);
                if (light_stratify) {
                    sampled_light_pos = a + ((j + u1) * (ab / (float)root_light_samples)) +
                        ((i + u2) * (ac / (float)root_light_samples));
                }
                else {
                    sampled_light_pos = a + u1 * ab + u2 * ac;
                }

                float3 shadow_ray_origin = attrib.intersection; //+ attrib.normal * cf.epsilon;
                float3 lightDir = normalize(sampled_light_pos - shadow_ray_origin);
                float light_dist = length(sampled_light_pos - shadow_ray_origin);
                ShadowPayload shadow_payload;
                shadow_payload.isVisible = true;

		if (k >= qlights.size()) {

		    for (int poggers = 0; poggers < 2; poggers++) {

			Triangle tri = poggers ? qlights[whichLight].tri1 : qlights[whichLight].tri2;

			float nDotWo = dot(tri.normal, -wi);

			float t = dot(tri.v1 - attrib.intersection, tri.normal) / dot(wi, tri.normal);
			float3 P = attrib.intersection + t * wi; // intersection in the object space

			float3 tmp0 = tri.v3 - tri.v1;
			float3 tmp1 = tri.v2 - tri.v1;
			float3 tmp2 = P - tri.v1;
			float tmp0dot0 = dot(tmp0, tmp0);
			float tmp0dot1 = dot(tmp0, tmp1);
			float tmp0dot2 = dot(tmp0, tmp2);
			float tmp1dot1 = dot(tmp1, tmp1);
			float tmp1dot2 = dot(tmp1, tmp2);
			float denom = tmp0dot0 * tmp1dot1 - tmp0dot1 * tmp0dot1;

			float u = (tmp1dot1 * tmp0dot2 - tmp0dot1 * tmp1dot2) / denom;
			float v = (tmp0dot0 * tmp1dot2 - tmp0dot1 * tmp0dot2) / denom;

			if (!(0 > u || u > 1 || 0 > v || v > 1 || u + v > 1 || nDotWo == 0.0f || t < 0.001)) {
			    lightDir = wi;
			    light_dist = t;
			    sampled_light_pos = P;
			    shadow_payload.isVisible = true;
			    break;
			}
			else {
			    shadow_payload.isVisible = false;
			}
		    }
		}
		if (!shadow_payload.isVisible) continue;

                Ray shadow_ray = make_Ray(shadow_ray_origin, lightDir, 1, cf.epsilon, light_dist - cf.epsilon);

                rtTrace(root, shadow_ray, shadow_payload);
                float3 n = attrib.normal;

                if (shadow_payload.isVisible) {
                    // rendering equation here: 
		    float3 brdf;
		    float pdf;
		    if (mv.brdf == 0) {
			brdf = (mv.diffuse / M_PIf) +
			    (mv.specular * ((mv.shininess + 2.0f) / (2.0f * M_PIf)) *
                            powf(fmaxf(dot(normalize(reflect(-attrib.wo, attrib.normal)), lightDir), .0f), mv.shininess));
			pdf = ((1.0f - t) * (fmaxf(dot(n, lightDir), .0f) / M_PIf)) + 
				t * ((mv.shininess + 1.0f) / (2.0f * M_PIf)) * 
				powf(fmaxf(dot(r, lightDir), .0f), mv.shininess);
		    }
		    if (mv.brdf == 1) {
			float lightDir_dot_n = dot(lightDir, n);
			float wo_dot_n = dot(attrib.wo, n);
			if (lightDir_dot_n > .0f && wo_dot_n > .0f) {
			    float t = fmaxf(0.25f, s_mean/(s_mean+d_mean));
			    float alpha = mv.roughness;
			    float3 h = normalize(lightDir + attrib.wo); // half angle: 
			    float theta_h = acosf(clamp(dot(h, n),0.0f,1.0f)); // not sure if need to clamp 0
			    // microfacet distribution function, D: 
			    float D = (alpha * alpha) / (M_PIf * powf(cosf(theta_h), 4.0f) *
			           powf((alpha * alpha) + powf(tanf(theta_h), 2.0f), 2.0f));
			    if (powf(cosf(theta_h), 4.0f)*(tanf(theta_h) == 0)) D = 0.0f;
			    // shadow-masking function, G:  
			    float G_1_lightDir = (lightDir_dot_n > .0f) ? 2.0f / (1.0f + sqrtf(1.0f + (alpha * alpha) *
				    powf(tanf(acosf(clamp(dot(lightDir, n),0.0f,1.0f))), 2.0f))) : .0f;
			    float G_1_wo = (wo_dot_n > .0f) ? 2.0f / (1.0f + sqrtf(1.0f + (alpha * alpha) *
				    powf(tanf(acosf(clamp(dot(attrib.wo, n),0.0f,1.0f))), 2.0f))) : .0f;
			    float G = G_1_lightDir * G_1_wo;

			    // fresnel function, F:
			    float3 F = mv.specular + (make_float3(1.0f) - mv.specular) *
				       powf(1.0f - dot(lightDir, h), 5.0f);
			    float3 f_brdf_GGX = (F * G * D) / (4.0f * lightDir_dot_n * wo_dot_n);
			    brdf = (mv.diffuse / M_PIf) + f_brdf_GGX;
			    pdf = fmaxf(((1 - t) * (lightDir_dot_n / M_PIf)) +
			     ((t * D * dot(n, h)) / (4.0f * fmaxf(dot(h, lightDir),0.0f))),0.0f);

		    }
		    else brdf = make_float3(0.0f); // assume f zero otherwise
		}

                    float3 x_prime = sampled_light_pos;
                    float3 x = shadow_ray_origin;
                    float3 n_light = normalize(cross(ab, ac));

                    float R = length(x - x_prime);
		    float pdf_nee;

                    // note: normal should point AWAY from the hitpoint, i.e. dot(n_light, x - x_prime) < 0
                    float G = (1.0f / powf(R, 2.0f)) *
		    fmaxf(dot(n, normalize(x_prime - x)), .0f) *
		    (fmaxf(dot(n_light, normalize(x_prime - x)), .0f));

		    if (cf.NEE == 2 && k < qlights.size() ) {
			pdf_nee = (R*R)/(area*dot(n_light, lightDir))/qlights.size();
			float weight = powf(pdf_nee,2.0f)/(powf(pdf_nee,2.0f) + powf(pdf,2.0f));
			sampled_result = weight * brdf * G * (1.0f/pdf_nee);
			weight = powf(pdf,2.0f)/(powf(pdf_nee,2.0f) + powf(pdf,2.0f));
			sampled_result += weight * brdf * G * (1.0f/pdf);
			sampled_result = sampled_result*100;
			//sampled_result = sampled_result/2;
		    }
		    else if (cf.NEE == 2) {
			pdf_nee = (R*R)/(area*dot(n_light, lightDir))/qlights.size();
			float weight = powf(pdf_nee,2.0f)/(powf(pdf_nee,2.0f) + powf(pdf,2.0f));
			sampled_result += weight * brdf * fmaxf(dot(n,wi),0) * (1.0f/pdf_nee);
			weight = powf(pdf,2.0f)/(powf(pdf_nee,2.0f) + powf(pdf,2.0f));
			sampled_result += weight * brdf * fmaxf(dot(n,wi),0) * (1.0f/pdf);
			//sampled_result = sampled_result/2;
		    }
		    else {
			sampled_result += brdf * G;
		    }

                }
            }
        }
        L_d += qlights[whichLight].color * sampled_result * (area / (float)light_samples);
    }


    
    if (cf.NEE && (payload.depth == 0)) {
        result += L_e;
        payload.radiance = (cf.NEE == 1 ? result + L_d : L_d) * payload.throughput;
    }
    else {
	if (cf.NEE == 2) {
	    result += L_e;
	}
        if (cf.NEE) {
	    result += L_d;
        }
        else {
            result += L_e;
        }
        payload.radiance = result * payload.throughput;
    }

    float q;
    if (cf.RR) {

        q = 1.0f - fmin(fmax(fmax(payload.throughput.x, payload.throughput.y), payload.throughput.z), 1.0f);
        // pick a num from 0 to 1, if less than q, terminate ray
        // i.e. make throughput 0
        if (rnd(payload.seed) < q) {
	    payload.done = true;
	    return;
        }
        else {
            bruh *= (1.0f / (1.0f - q));
        }
    }

    payload.throughput *= bruh;
    payload.origin = attrib.intersection;
    payload.dir = wi;
    payload.depth++;
}
