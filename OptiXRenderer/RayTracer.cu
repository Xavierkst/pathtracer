#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );
rtDeclareVariable(ShadowPayload, shadowPayload, rtPayload, );
rtDeclareVariable(float1, t, rtIntersectionDistance, );
rtDeclareVariable(float3, eye, , );

RT_PROGRAM void closestHit()
{
    // TODO: calculate the color using the Blinn-Phong reflection model

    //float3 result = make_float3(0, 1, 0);
    float3 result = attrib.ambient + attrib.emission;
     
    // QUESTION: is the HALF-ANGLE: H = normalize(L + V), where L is the direction from hitpoint to light, and V is dir from hitPt to eye?
    //for (int i = 0; i < plights.size(); i++) {
    //    float3 half_angle = normalize((plights[i].light_pos - payload.hitPoint) + (eye - payload.hitPoint));

    //    result +=  plights[i].light_color / 
    //        (plights[i].attenuation.constant + plights[i].attenuation.linear + 
    //            plights[i].attenuation.quadratic) * (attrib.diffuse * fmaxf(dot(payload.hitPointNormal, (plights[i].light_pos - payload.hitPoint)), .0f) + 
    //                attrib.specular * pow(fmaxf(dot(payload.hitPointNormal, half_angle), .0f), attrib.shininess)); ;
    //}
    //for (int i = 0; i < dlights.size(); i++) {
    //    float3 half_angle = normalize(-dlights[i].light_dir + (eye - payload.hitPoint));

    //    result += dlights[i].light_color /
    //        (dlights[i].attenuation.constant + dlights[i].attenuation.linear +
    //            dlights[i].attenuation.quadratic) * (attrib.diffuse * fmaxf(dot(payload.hitPointNormal, -dlights[i].light_dir), .0f) + 
    //                attrib.specular * pow(fmaxf(dot(payload.hitPointNormal, half_angle), .0f), attrib.shininess));
    //    //result += 
    //}
    //result += shadowPayload.isVisible * result;
    payload.radiance = result;
}