#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );
rtDeclareVariable(intersectionData, intersectData, attribute intersectData, );
//rtDeclareVariable(ShadowPayload, shadowPayload, rtPayload, );
rtDeclareVariable(float1, t, rtIntersectionDistance, );
rtDeclareVariable(float3, eye, , );
// depth from Renderer.cpp
rtDeclareVariable(int, depth, , );

RT_PROGRAM void closestHit()
{
    // TODO: calculate the color using the Blinn-Phong reflection model
    float epsilon = .001f;
    float3 result = attrib.ambient + attrib.emission;
    //printf("%f, %f, %f\n", attrib.ambient.x, attrib.ambient.y, attrib.ambient.z);
    //float3 result = attrib.ambient;
     
    // QUESTION: is the HALF-ANGLE: H = normalize(L + V), where L is the direction from hitpoint to light, and V is dir from hitPt to eye?
    for (int i = 0; i < plights.size(); i++) {
        //cast shadow ray
        float3 lightVec = (plights[i].light_pos - intersectData.hitPoint);
        float distToLight = length(lightVec);
        lightVec = normalize(lightVec);

        // create shadow ray and cast it
    	Ray shadowRay = make_Ray(intersectData.hitPoint, lightVec, 1, epsilon, distToLight);
        ShadowPayload shadowPayload; 
        shadowPayload.isVisible = true;
    	rtTrace(root, shadowRay, shadowPayload); 

        //rtPrintf("%d", shadowPayload.isVisible);

        float3 half_angle = normalize(lightVec + normalize(intersectData.rayOrig - intersectData.hitPoint));
        //float3 half_angle = normalize((-intersectData.rayDir) + (eye - intersectData.hitPoint));

        if (shadowPayload.isVisible) {
            //result = (half_angle);
            result += (plights[i].light_color /
                (plights[i].attenuation.constant + plights[i].attenuation.linear * distToLight +
                    plights[i].attenuation.quadratic * powf(distToLight, 2.0f))) * 
                (attrib.diffuse * fmaxf(dot(intersectData.hitPointNormal,
                        normalize(lightVec)), .0f) +
                        attrib.specular * powf(fmaxf(dot( intersectData.hitPointNormal, half_angle), .0f),
                            attrib.shininess));
        }
    }

    for (int i = 0; i < dlights.size(); i++) {
        //cast shadow ray
    	float distToLight = RT_DEFAULT_MAX;
    	Ray shadowRay = make_Ray(intersectData.hitPoint, -dlights[i].light_dir, 1, epsilon, distToLight);
        ShadowPayload shadowPayload; 
        shadowPayload.isVisible = true;
    	rtTrace(root, shadowRay, shadowPayload); 

        float3 half_angle = normalize(normalize(-dlights[i].light_dir) + (intersectData.rayOrig - intersectData.hitPoint));
        //float3 half_angle = normalize(-intersectData.rayDir + (eye - intersectData.hitPoint));
        if (shadowPayload.isVisible) {
            result += (dlights[i].light_color /
                (dlights[i].attenuation.constant + dlights[i].attenuation.linear /** distToLight*/ +
                    dlights[i].attenuation.quadratic /** powf(distToLight, 2.0f)*/)) * (attrib.diffuse * fmaxf(
                        dot(intersectData.hitPointNormal, normalize(-dlights[i].light_dir)), .0f) +
                        attrib.specular * powf(fmaxf(dot(intersectData.hitPointNormal, half_angle), .0f),
                            attrib.shininess));
        }
    }

    // pass the new ray dir and reflection dir into payload 
    // to be used in rayGeneration do-While loop: 
    payload.rayOrigin = intersectData.hitPoint /*+ epsilon * intersectData.hitPointNormal*/;
    payload.rayDir = intersectData.reflectDir;

    if (payload.depth == depth) {
        payload.radiance = result;
        payload.spec = attrib.specular;
    }
    //else payload.radiance = pow(attrib.specular, (depth - (payload.depth + 1)));
    else {
        //float specularExp = depth - (payload.depth + 1); 
        //payload.radiance = make_float3(powf(attrib.specular.x, specularExp), 
        //    powf(attrib.specular.y, specularExp),
        //    powf(attrib.specular.z, specularExp))  * result;
        payload.radiance = result;
        payload.spec *= attrib.specular;
    }
    --payload.depth;
}