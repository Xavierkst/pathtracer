#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"
#include "Config.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;
rtBuffer<QuadLight> qlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

rtBuffer<Config> config; // Config

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

rtDeclareVariable(uint, light_samples, , );
rtDeclareVariable(uint, light_stratify, , );
rtDeclareVariable(uint, next_event_est, , );
rtDeclareVariable(uint, sampling_method, , );

RT_PROGRAM void closestHit()
{
    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float3 result = mv.ambient + mv.emission;

    // Calculate the direct illumination of point lights
    for (int i = 0; i < plights.size(); i++)
    {
        // Shoot a shadow to determin whether the object is in shadow
        float3 lightDir = normalize(plights[i].location - attrib.intersection);
        float lightDist = length(plights[i].location - attrib.intersection);
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;
        Ray shadowRay = make_Ray(attrib.intersection + lightDir * cf.epsilon, 
            lightDir, 1, cf.epsilon, lightDist);
        rtTrace(root, shadowRay, shadowPayload);

        // If not in shadow
        if (shadowPayload.isVisible)
        {
            float3 H = normalize(lightDir + attrib.wo);
            float att = dot(plights[i].attenuation, make_float3(1, lightDist, lightDist * lightDist));
            float3 I = mv.diffuse * fmaxf(dot(attrib.normal, lightDir), 0);
            I += mv.specular * pow(fmaxf(dot(attrib.normal, H), 0), mv.shininess);
            I *= plights[i].color / att;
            result += I;
        }
    }

    // Calculate the direct illumination of directional lights
    for (int i = 0; i < dlights.size(); i++)
    {
        // Shoot a shadow to determin whether the object is in shadow
        float3 lightDir = dlights[i].direction;
        float lightDist = RT_DEFAULT_MAX;
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;
        Ray shadowRay = make_Ray(attrib.intersection + lightDir * cf.epsilon, 
            lightDir, 1, cf.epsilon, lightDist);
        rtTrace(root, shadowRay, shadowPayload);

        // If not in shadow
        if (shadowPayload.isVisible)
        {
            float3 H = normalize(lightDir + attrib.wo);
            float3 I = mv.diffuse * fmaxf(dot(attrib.normal, lightDir), 0);
            I += mv.specular * pow(fmaxf(dot(attrib.normal, H), 0), mv.shininess);
            I *= dlights[i].color;
            result += I;
        }
    }

    // Compute the final radiance
    payload.radiance = result * payload.throughput;

    // Calculate reflection
    if (length(mv.specular) > 0)
    {
        // Set origin and dir for tracing the reflection ray
        payload.origin = attrib.intersection;
        payload.dir = reflect(-attrib.wo, attrib.normal); // mirror reflection

        payload.depth++;
        payload.throughput *= mv.specular;
    }
    else
    {
        payload.done = true;
    }
}

RT_PROGRAM void analyticDirect() {

    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float3 result = mv.ambient + mv.emission;

    // for-loop here to calculate contribution of quadLights
    // if no light samples, do analytical direct:
    if (light_samples == 0) {
        for (int i = 0; i < qlights.size(); ++i) {
            float3 f_brdf = mv.diffuse / M_PIf;// brdf function 
            float3 hitPt = attrib.intersection;
            float3 hitPtNormal = attrib.normal;

            float3 a = qlights[i].tri1.v1;
            float3 b = qlights[i].tri1.v2;
            float3 c = qlights[i].tri2.v2;
            float3 d = qlights[i].tri1.v3;

            float3 points[] = { qlights[i].tri1.v1, qlights[i].tri1.v2, qlights[i].tri2.v2, qlights[i].tri1.v3 };

            float3 p1 = points[0]; float3 p2 = points[1]; float3 p3 = points[2]; float3 p4 = points[3];
            float theta_1 = acosf(dot(normalize(p1 - hitPt), normalize(p2 - hitPt)));
            float theta_2 = acosf(dot(normalize(p2 - hitPt), normalize(p3 - hitPt)));
            float theta_3 = acosf(dot(normalize(p3 - hitPt), normalize(p4 - hitPt)));
            float theta_4 = acosf(dot(normalize(p4 - hitPt), normalize(p1 - hitPt)));

            float3 gamma_1 = normalize(cross((p1 - hitPt), (p2 - hitPt)));
            float3 gamma_2 = normalize(cross((p2 - hitPt), (p3 - hitPt)));
            float3 gamma_3 = normalize(cross((p3 - hitPt), (p4 - hitPt)));
            float3 gamma_4 = normalize(cross((p4 - hitPt), (p1 - hitPt)));

            float3 irradiance_vec = 0.5f * ((theta_1 * gamma_1) +
                (theta_2 * gamma_2) + (theta_3 * gamma_3) + (theta_4 * gamma_4));

            float3 dir_radiance = f_brdf * qlights[i].color * dot(irradiance_vec, hitPtNormal);
            result += dir_radiance;
        }
    }

    payload.radiance = result;

    payload.done = true;
}


RT_PROGRAM void direct() {

    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float3 result = mv.ambient + mv.emission;

    for (int k = 0; k < qlights.size(); ++k) {
        float3 sampled_result = make_float3(.0f);
        // Compute direct lighting equation for w_i_k ray, for k = 1 to N*N
        float3 a = qlights[k].tri1.v1;
        float3 b = qlights[k].tri1.v2;
        float3 c = qlights[k].tri2.v3;
        float3 d = qlights[k].tri2.v2;

        float3 ac = c - a;
        float3 ab = b - a;
        float area = length(cross(ab, ac));
        int root_light_samples = (int)sqrtf(light_samples);
        // check if stratify or random sampling
        // double for loop here 
        for (int i = 0; i < root_light_samples; ++i) {
            for (int j = 0; j < root_light_samples; ++j) {
                // generate random float vals u1 and u2
                float u1 = rnd(payload.seed);
                float u2 = rnd(payload.seed);

                float3 sampled_light_pos;
                if (light_stratify) {
                    sampled_light_pos = a + ((j + u1) * (ab / (float)root_light_samples)) +
                        ((i + u2) * (ac / (float)root_light_samples));
                }
                else {
                    sampled_light_pos = a + u1 * ab + u2 * ac;
                }
                float3 shadow_ray_origin = attrib.intersection /*+ attrib.normal * cf.epsilon*/;
                float3 shadow_ray_dir = normalize(sampled_light_pos - shadow_ray_origin);
                float light_dist = length(sampled_light_pos - shadow_ray_origin);
                Ray shadow_ray = make_Ray(shadow_ray_origin, shadow_ray_dir, 1, cf.epsilon, light_dist - cf.epsilon);

                ShadowPayload shadow_payload;
                shadow_payload.isVisible = true;
                rtTrace(root, shadow_ray, shadow_payload);

                if (shadow_payload.isVisible) {
                    // rendering equation here: 
                    //float3 w_i = sampled_light_pos;
                    float3 f_brdf = (mv.diffuse / M_PIf) +
                        (mv.specular * ((mv.shininess + 2.0f) / (2.0f * M_PIf)) *
                            powf(fmaxf(dot(normalize(reflect(-attrib.wo, attrib.normal)), normalize(sampled_light_pos - shadow_ray_origin)), .0f), mv.shininess));

                    float3 x_prime = sampled_light_pos;
                    float3 x = shadow_ray_origin;
                    float3 n = attrib.normal;
                    //float3 n_light = normalize(qlights[k].tri1.normal);
                    float3 n_light = normalize(cross(ab, ac));
                    //n_light = dot(n_light, normalize(x_prime - x)) > .0f ? n_light : -n_light;

                    float R = length(x - x_prime);

                    // note: normal should point AWAY from the hitpoint, i.e. dot(n_light, x - x_prime) < 0
                    float G = (1.0f / powf(R, 2.0f)) * fmaxf(dot(n, normalize(x_prime - x)), .0f) *
                        (fmaxf(dot(n_light, normalize(x_prime - x)), .0f));

                    sampled_result += f_brdf * G;
                }
            }
        }
        result += qlights[k].color * sampled_result * (area / (float)light_samples);
    }
    //rtPrintf("throughput val: %f \n", payload.throughput);
    payload.radiance = result;

    payload.done = true;
}

RT_PROGRAM void pathTracer() {

    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float3 n = normalize(attrib.normal);
    float3 L_e = mv.emission;
    float3 result = make_float3(.0f);
    float3 L_d = make_float3(.0f);
    float3 K_s = mv.specular;
    float3 K_d = mv.diffuse;
    float3 r = normalize(reflect(-attrib.wo, attrib.normal));

    float exp_beta = 2.0f;

    // variables for direct lighting
    int root_light_samples = (int)sqrtf(light_samples);
    float pdf_NEE = .0f;

    // When next event estimation is ON:
    // if an indir ray ever strikes light source (and it is NOT the first ray cast)
    // ray should be terminated
    if (cf.next_event_est && attrib.objType == LIGHT && payload.depth != 0) {
        payload.depth = cf.maxDepth;
        payload.done = true;
        payload.radiance = result;
        return;
    }

    // Add indirect lighting here:
    // generate randomize ray direction w_i
    float zeta_0 = rnd(payload.seed);
    float zeta_1 = rnd(payload.seed);
    float zeta_2 = rnd(payload.seed);
    float K_s_avg = (K_s.x + K_s.y + K_s.z) / 3.0f;
    float K_d_avg = (K_d.x + K_d.y + K_d.z) / 3.0f;
    //rtPrintf("ks %f\n", K_s_avg);
    //rtPrintf("kd %f\n", K_d_avg);
    float t = .0f;
    // make sure denom not 0, else check if phong or ggx
    // and set t accordingly
    if ((K_s_avg == .0f) && (K_d_avg == .0f)) {
        //rtPrintf("heeeere\n");
        // if denom is 0, t is 0 when brdf is modified phong, and 1 for ggx
        if (mv.brdf_type == MOD_PHONG)
            t = .0f;
        else if (mv.brdf_type == GGX)
            t = 1.0f;
    }
    else {
        // denom is not zero
        //rtPrintf("OR heeeere\n");
        //rtPrintf("ks %f\n", K_s_avg);
        //rtPrintf("kd %f\n", K_d_avg);
        float k_val = K_s_avg / (K_s_avg + K_d_avg);
        if (mv.brdf_type == MOD_PHONG) {
            t = k_val;
            //rtPrintf("k value is: %f\n", k_val);
            //if (t != .0f) 
            //    rtPrintf("t value is: %f\n", t);

        }
        else {
            t = fmaxf(.25f, k_val);
            //rtPrintf("t value is: %f\n", t);
        }
    }

    float theta = .0f;
    float phi = .0f;
    switch (sampling_method) {
    case HEMISPHERE_SAMPLING:
        //rtPrintf("hemisphere here\n");
        phi = 2.0f * M_PIf * zeta_2;
        theta = acosf(zeta_1);
        break;
    case COSINE_SAMPLING:
        //rtPrintf("cosine here\n");
        phi = 2.0f * M_PIf * zeta_2;
        theta = acosf(sqrtf(zeta_1));
        break;
    case BRDF_SAMPLING:
        //rtPrintf("brdf here\n");
        // phi remains the same for either specular or diffuse pdf
        phi = 2.0f * M_PIf * zeta_2;
        if (zeta_0 > t)
            theta = acosf(sqrtf(zeta_1)); // theta_diffuse
        else
            theta = acosf(powf(zeta_1, (1.0f / (mv.shininess + 1.0f)))); // theta_specular
        break;
    default:
    }

    // qn: why rotate s wrt the z-axis? and not the y-axis?
    float3 sample_s = make_float3(cosf(phi) * sinf(theta), sinf(phi) * sinf(theta), cosf(theta));

    // generate coordinate frame at the intersect point
    //float3 n = normalize(attrib.normal);
    float3 w = n;
    // if specular brdf chosen, center sample_s at reflection vector r
    if ((zeta_0 <= t) && (sampling_method == BRDF_SAMPLING)) w = r;

    //else w = normalize(n);

    float3 a = make_float3(.0f, 1.0f, .0f);
    // incase a and w are closely aligned, swap a out for 
    // a diff arbitrary vector <1,0,0> instead of <0,1,0>
    if (1.0f - fabsf(dot(a, w)) <= 1.0f) {
        a = make_float3(1.0f, .0f, .0f);
    }

    float3 u = normalize(cross(a, w));
    float3 v = normalize(cross(w, u)); // i dont think need to normalize

    float theta_h_sample = atanf((mv.roughness * sqrtf(zeta_1)) / sqrtf(1.0f - zeta_1));
    float phi_h_sample = 2.0f * M_PIf * zeta_2;
    float3 h_sample = make_float3(cosf(phi_h_sample) * sinf(theta_h_sample),
        sinf(phi_h_sample) * sinf(theta_h_sample),
        cosf(theta_h_sample));

    float3 w_h = n;
    float3 a_h = make_float3(.0f, 1.0f, .0f);
    if (1.0f - fabsf(dot(a_h, w_h)) <= 1.0f) {
        a_h = make_float3(1.0f, .0f, .0f);
    }

    float3 u_h = normalize(cross(a_h, w_h));
    float3 v_h = normalize(cross(w_h, u_h));

    // rotate h:
    float3 w_i = make_float3(.0f);
    // get randomized new ray dir -- choose a sampling method
    if (mv.brdf_type == MOD_PHONG)
        w_i = normalize((sample_s.x * u + sample_s.y * v + sample_s.z * w));
    else {
        h_sample = (h_sample.x * u_h + h_sample.y * v_h + h_sample.z * w_h);
        w_i = normalize(reflect(-attrib.wo, h_sample));
    }

    if (cf.next_event_est) {
        // Add direct lighting here:
        for (int k = 0; k < qlights.size(); ++k) {
            float3 sampled_result = make_float3(.0f);
            float pdf_lights_k = .0f;
            float pdf_brdf = .0f;
            float3 w_i_dir = make_float3(.0f);
            float3 h = make_float3(.0f);
            float D = .0f;
            // Compute direct lighting equation for w_i_k ray, for k = 1 to N*N
            float3 a = qlights[k].tri1.v1;
            float3 b = qlights[k].tri1.v2;
            float3 c = qlights[k].tri2.v3;
            float3 d = qlights[k].tri2.v2;

            float3 ac = c - a;
            float3 ab = b - a;
            float area = length(cross(ab, ac));
            int root_light_samples = (int)sqrtf(light_samples);
            // check if stratify or random sampling
            // double for loop here 
            for (int i = 0; i < root_light_samples; ++i) {
                for (int j = 0; j < root_light_samples; ++j) {
                    // generate random float vals u1 and u2
                    float u1 = rnd(payload.seed);
                    float u2 = rnd(payload.seed);

                    float3 sampled_light_pos;
                    if (light_stratify) {
                        sampled_light_pos = a + ((j + u1) * (ab / (float)root_light_samples)) +
                            ((i + u2) * (ac / (float)root_light_samples));
                    }
                    else {
                        sampled_light_pos = a + u1 * ab + u2 * ac;
                    }

                    float3 shadow_ray_origin = attrib.intersection /*+ attrib.normal * cf.epsilon*/;
                    float3 shadow_ray_dir = normalize(sampled_light_pos - shadow_ray_origin);
                    float light_dist = length(sampled_light_pos - shadow_ray_origin);
                    Ray shadow_ray = make_Ray(shadow_ray_origin, shadow_ray_dir, 1, cf.epsilon, light_dist - cf.epsilon);

                    ShadowPayload shadow_payload;
                    shadow_payload.isVisible = true;
                    rtTrace(root, shadow_ray, shadow_payload);

                    if (shadow_payload.isVisible) {
                        //float D = .0f;

                        // rendering equation here: 
                        //float3 w_i = sampled_light_pos;
                        w_i_dir = normalize(sampled_light_pos - shadow_ray_origin);

                        float3 f_brdf = make_float3(.0f);
                        if (mv.brdf_type == MOD_PHONG) {
                            f_brdf = (mv.diffuse / M_PIf) +
                                (mv.specular * ((mv.shininess + 2.0f) / (2.0f * M_PIf)) *
                                    powf(fmaxf(dot(r, w_i_dir), .0f), mv.shininess));
                        }
                        else {
                            float wi_dot_n_dir = dot(w_i_dir, n);
                            float wo_dot_n_dir = dot(attrib.wo, n);
                            if (wi_dot_n_dir > .0f && wo_dot_n_dir > .0f) {
                                float alpha = mv.roughness;
                                float3 h = normalize(w_i_dir + attrib.wo); // half angle: 
                                float theta_h = acosf(dot(h, n)); // not sure if need to clamp 0
                                float cos_theta_h_4 = powf(cosf(theta_h), 4.0f);
                                float alpha_tan_theta_h_sq = (alpha * alpha) + powf(tanf(theta_h), 2.0f);
                                // make sure denom not 0, else set D to 0
                                if (cos_theta_h_4 * alpha_tan_theta_h_sq != .0f) D = (alpha * alpha) / (M_PIf * cos_theta_h_4 * powf(alpha_tan_theta_h_sq, 2.0f));

                                // microfacet distribution function, D: 
                                //float D = (alpha * alpha) / (M_PIf * powf(cosf(theta_h), 4.0f) * powf((alpha * alpha) + powf(tanf(theta_h), 2.0f), 2.0f));

                                // shadow-masking function, G:  
                                float G_1_wi = (wi_dot_n_dir > .0f) ? 2.0f / (1.0f + sqrtf(1.0f + (alpha * alpha) * powf(tanf(acosf(dot(w_i_dir, n))), 2.0f))) : .0f;
                                float G_1_wo = (wo_dot_n_dir > .0f) ? 2.0f / (1.0f + sqrtf(1.0f + (alpha * alpha) * powf(tanf(acosf(dot(attrib.wo, n))), 2.0f))) : .0f;
                                float G = G_1_wi * G_1_wo;
                                // fresnel function, F:
                                float3 F = K_s + (1.0f - K_s) * powf(1.0f - dot(w_i_dir, h), 5.0f);
                                float3 f_brdf_GGX = (F * G * D) / (4.0f * wi_dot_n_dir * wo_dot_n_dir);

                                f_brdf = (K_d / M_PIf) + f_brdf_GGX;
                            }
                        }

                        float3 x_prime = sampled_light_pos;
                        float3 x = shadow_ray_origin;
                        float3 n = attrib.normal;
                        float3 n_light = normalize(cross(ab, ac));

                        float R = length(x - x_prime);

                        // note: normal should point AWAY from the hitpoint, i.e. dot(n_light, x - x_prime) < 0
                        float G = (1.0f / powf(R, 2.0f)) * fmaxf(dot(n, normalize(x_prime - x)), .0f) *
                            (fmaxf(dot(n_light, normalize(x_prime - x)), .0f));

                        sampled_result += f_brdf * G;
                        //pdf_lights_k += (powf(R, 2.0f) / (area * fabsf(dot(n, w_i_dir)))); 
                    }
                }
            }
            // calculate weight Wi for this given w_i generated
            pdf_NEE = pdf_lights_k /** (1.0f / (float) qlights.size())*/;

            // calculate pdf_brdf 
            pdf_brdf = ((1.0f - t) * fmaxf(dot(n, w_i_dir), .0f) / M_PIf) + ((t * D * dot(n, h)) / (4.0f * dot(h, w_i_dir)));
         
            // calc power heuristic: 
            float pdf_denom_sum = powf(pdf_NEE, exp_beta) + powf(pdf_brdf, exp_beta);
            float pdf_numerator = powf(pdf_NEE, exp_beta);
            float weight_i = pdf_numerator / pdf_denom_sum;
        
            if (cf.next_event_est == MIS) {
                //rtPrintf("here");
                L_d += qlights[k].color * sampled_result * (area / (float)light_samples) * (1.0f / pdf_NEE) * weight_i;
            }
            else {
                // divide brdf by the pdf here
                L_d += qlights[k].color * sampled_result * (area / (float)light_samples);
            }
        }
    }

    //// Add indirect lighting here:
    //// generate randomize ray direction w_i
    //float zeta_0 = rnd(payload.seed);
    //float zeta_1 = rnd(payload.seed);
    //float zeta_2 = rnd(payload.seed);
    //float K_s_avg = (K_s.x + K_s.y + K_s.z) / 3.0f;
    //float K_d_avg = (K_d.x + K_d.y + K_d.z) / 3.0f;
    ////rtPrintf("ks %f\n", K_s_avg);
    ////rtPrintf("kd %f\n", K_d_avg);
    //float t = .0f;
    //// make sure denom not 0, else check if phong or ggx
    //// and set t accordingly
    //if ((K_s_avg == .0f) && (K_d_avg == .0f)) {
    //    //rtPrintf("heeeere\n");
    //    // if denom is 0, t is 0 when brdf is modified phong, and 1 for ggx
    //    if (mv.brdf_type == MOD_PHONG) 
    //        t = .0f;
    //    else if (mv.brdf_type == GGX) 
    //        t = 1.0f;
    //}
    //else {
    //    // denom is not zero
    //    //rtPrintf("OR heeeere\n");
    //    //rtPrintf("ks %f\n", K_s_avg);
    //    //rtPrintf("kd %f\n", K_d_avg);
    //    float k_val = K_s_avg / (K_s_avg + K_d_avg);
    //    if (mv.brdf_type == MOD_PHONG) {
    //        t = k_val;
    //        //rtPrintf("k value is: %f\n", k_val);
    //        //if (t != .0f) 
    //        //    rtPrintf("t value is: %f\n", t);

    //    }
    //    else {
    //        t = fmaxf(.25f, k_val);
    //        //rtPrintf("t value is: %f\n", t);
    //    }
    //}

    //float theta = .0f;
    //float phi = .0f;
    //switch (sampling_method) {
    //    case HEMISPHERE_SAMPLING: 
    //        //rtPrintf("hemisphere here\n");
    //        phi = 2.0f * M_PIf * zeta_2;
    //        theta = acosf(zeta_1);
    //        break;
    //    case COSINE_SAMPLING: 
    //        //rtPrintf("cosine here\n");
    //        phi = 2.0f * M_PIf * zeta_2;
    //        theta = acosf(sqrtf(zeta_1));
    //        break;
    //    case BRDF_SAMPLING: 
    //        //rtPrintf("brdf here\n");
    //        // phi remains the same for either specular or diffuse pdf
    //        phi = 2.0f * M_PIf * zeta_2;
    //        if (zeta_0 > t) 
    //            theta = acosf(sqrtf(zeta_1)); // theta_diffuse
    //        else 
    //            theta = acosf(powf(zeta_1, (1.0f / (mv.shininess + 1.0f)))); // theta_specular
    //        break;
    //    default: 
    //        break;
    //}

    //// qn: why rotate s wrt the z-axis? and not the y-axis?
    //float3 sample_s = make_float3(cosf(phi) * sinf(theta), sinf(phi) * sinf(theta), cosf(theta));
    //
    //// generate coordinate frame at the intersect point
    //float3 w = n;
    //// if specular brdf chosen, center sample_s at reflection vector r
    //if ((zeta_0 <= t) && (sampling_method == BRDF_SAMPLING)) w = r;

    ////else w = normalize(n);

    //float3 a = make_float3(.0f, 1.0f, .0f);
    //// incase a and w are closely aligned, swap a out for 
    //// a diff arbitrary vector <1,0,0> instead of <0,1,0>
    //if (1.0f - fabsf(dot(a, w)) <= 1.0f) {
    //    a = make_float3(1.0f, .0f, .0f);
    //}

    //float3 u = normalize(cross(a, w));
    //float3 v = normalize(cross(w, u)); // i dont think need to normalize

    //float theta_h_sample = atanf((mv.roughness * sqrtf(zeta_1)) / sqrtf(1.0f - zeta_1));
    //float phi_h_sample = 2.0f * M_PIf * zeta_2;
    //float3 h_sample = make_float3(cosf(phi_h_sample) * sinf(theta_h_sample), 
    //    sinf(phi_h_sample)*sinf(theta_h_sample), 
    //        cosf(theta_h_sample));

    //float3 w_h = n; 
    //float3 a_h = make_float3(.0f, 1.0f, .0f); 
    //if (1.0f - fabsf(dot(a_h, w_h)) <= 1.0f) {
    //    a_h = make_float3(1.0f, .0f, .0f);
    //}
    //
    //float3 u_h = normalize(cross(a_h, w_h)); 
    //float3 v_h = normalize(cross(w_h, u_h));

    //// rotate h:
    //float3 w_i = make_float3(.0f); 
    //// get randomized new ray dir -- choose a sampling method
    //if (mv.brdf_type == MOD_PHONG) 
    //    w_i = normalize((sample_s.x * u + sample_s.y * v + sample_s.z * w));
    //else {
    //    h_sample = (h_sample.x * u_h + h_sample.y * v_h + h_sample.z * w_h);
    //    w_i = normalize(reflect(-attrib.wo, h_sample));
    //}



    // calculate the summation of all pdfs here (i.e. pdf_nee + pdf_brdf)
    // using h, w_i, 
    
    // the BRDF 
    float3 f_brdf = make_float3(0.0f);
    float pdf = 1.0f;
    float3 addon_throughput = make_float3(.0f);

    switch (sampling_method) {
    case HEMISPHERE_SAMPLING:
        //rtPrintf("hemisphere here");
        f_brdf = (mv.diffuse / M_PIf) +
            (mv.specular * ((mv.shininess + 2.0f) / (2.0f * M_PIf)) *
                powf(fmaxf(dot(r, w_i), .0f), mv.shininess));
        pdf = 1.0f / (2.0f * M_PIf);
        addon_throughput = (f_brdf * fmaxf(dot(n, w_i), .0f) * (1.0f / pdf));
        break;
    case COSINE_SAMPLING:
        //rtPrintf("cosine here");
        f_brdf = (mv.diffuse / M_PIf) +
            (mv.specular * ((mv.shininess + 2.0f) / (2.0f * M_PIf)) *
                powf(fmaxf(dot(r, w_i), .0f), mv.shininess));
        pdf = fmaxf(dot(n, w_i), .0f) / (M_PIf);
        addon_throughput = (f_brdf)*fmaxf(dot(n, w_i), .0f) * (1.0f / pdf);
        break;
    case BRDF_SAMPLING:
        float pdf_NEE_brdf = .0f;
        //rtPrintf("brdf here");
        // check the material whether to use mod-phong or GGX brdf
        if (mv.brdf_type == MOD_PHONG) {
            f_brdf = (K_d / M_PIf) +
                (K_s * ((mv.shininess + 2.0f) / (2.0f * M_PIf)) *
                    powf(fmaxf(dot(r, w_i), .0f), mv.shininess));

            pdf = ((1.0f - t) * (dot(n, w_i) / M_PIf)) +
                t * ((mv.shininess + 1.0f) / (2.0f * M_PIf)) *
                powf(fmaxf(dot(r, w_i), .0f), mv.shininess);
        }
        else {
            // construct GGX BRDF: 
            float pdf_lights_k = .0f;

            float wi_dot_n = dot(w_i, n);
            float wo_dot_n = dot(attrib.wo, n);
            if (wi_dot_n > .0f && wo_dot_n > .0f) {
                float alpha = mv.roughness;
                float3 h = normalize(w_i + attrib.wo); // half angle: 
                float theta_h = acosf(dot(h, n)); // not sure if need to clamp 0
                // microfacet distribution function, D: 
                float cos_theta_h_4 = powf(cosf(theta_h), 4.0f);
                float alpha_tan_theta_h_sq = (alpha * alpha) + powf(tanf(theta_h), 2.0f);
                float D = .0f;
                // make sure denom not 0, else set D to 0
                if (cos_theta_h_4 * alpha_tan_theta_h_sq != .0f) D = (alpha * alpha) / (M_PIf * cos_theta_h_4 * powf(alpha_tan_theta_h_sq, 2.0f));
                //D = (alpha * alpha) / (M_PIf * powf(cosf(theta_h), 4.0f) * powf(alpha_tan_theta_h_sq, 2.0f));
            // shadow-masking function, G:  
                float G_1_wi = (wi_dot_n > .0f) ? 2.0f / (1.0f + sqrtf(1.0f + (alpha * alpha) * powf(tanf(acosf(dot(w_i, n))), 2.0f))) : .0f;
                float G_1_wo = (wo_dot_n > .0f) ? 2.0f / (1.0f + sqrtf(1.0f + (alpha * alpha) * powf(tan(acosf(dot(attrib.wo, n))), 2.0f))) : .0f;
                float G = G_1_wi * G_1_wo;
                // fresnel function, F:
                float3 F = K_s + (1.0f - K_s) * powf(1.0f - fmaxf(dot(w_i, h), .0f), 5.0f);
                float3 f_brdf_GGX = (F * G * D) / (4.0f * wi_dot_n * wo_dot_n);
                f_brdf = (K_d / M_PIf) + f_brdf_GGX;
                pdf = ((1.0f - t) * fmaxf(dot(n, w_i), .0f) / M_PIf) + ((t * D * dot(n, h)) / (4.0f * dot(h, w_i)));

                for (int k = 0; k < qlights.size(); ++k) {
                    //float3 sampled_result = make_float3(.0f);
                    float pdf_brdf = .0f;
                    // Compute direct lighting equation for w_i_k ray, for k = 1 to N*N
                    float3 a = qlights[k].tri1.v1;
                    float3 b = qlights[k].tri1.v2;
                    float3 c = qlights[k].tri2.v3;
                    float3 d = qlights[k].tri2.v2;

                    float3 ac = c - a;
                    float3 ab = b - a;
                    float area = length(cross(ab, ac));


                    float3 shadow_ray_origin = attrib.intersection /*+ attrib.normal * cf.epsilon*/;
                    float3 shadow_ray_dir = w_i;
                    // trace the ray and see if it hits a light source
                    Ray shadow_ray = make_Ray(shadow_ray_origin, shadow_ray_dir, 1, cf.epsilon, RT_DEFAULT_MAX);
                    //float light_dist = length(sampled_light_pos - shadow_ray_origin);

                    ShadowPayload shadow_payload;
                    shadow_payload.isVisible = true;
                    rtTrace(root, shadow_ray, shadow_payload);

                    if (!shadow_payload.isVisible && shadow_payload.objType == LIGHT) {
                        float3 x_prime = shadow_payload.intersectPt;
                        float3 x = shadow_ray_origin;
                        float3 n_light = normalize(cross(ab, ac));
                        float R = length(x - x_prime);

                        pdf_lights_k += (powf(R, 2.0f) / (area * fabsf(dot(n, w_i))));
                    }
                    /*else {
                        pdf_lights_k += .0f;
                    }*/
                }

            }
            else f_brdf = make_float3(.0f); // assume f zero otherwise
            pdf_NEE_brdf = /*(1.0f / qlights.size()) **/ pdf_lights_k;
        }

        float pdf_denom_sum = powf(pdf_NEE_brdf, exp_beta) + powf(pdf, exp_beta);
        float pdf_numerator = powf(pdf, exp_beta);
        float weight_i_brdf = pdf_numerator / pdf_denom_sum;

        if (cf.next_event_est == MIS) {
            addon_throughput = (f_brdf * fmaxf(dot(n, w_i), .0f) * (1.0f / pdf) * weight_i_brdf);
        }
        else {
            addon_throughput = (f_brdf * fmaxf(dot(n, w_i), .0f) * (1.0f / pdf));
        }
        break;
        default:
    }


    // Check if its first intersected surface
    if (cf.next_event_est && (payload.depth == 0)) {
        result += L_e;
        payload.radiance = (result + L_d) * payload.throughput;
    }
    // on the last bounce, we return only emission term
    // NEE true (1) or false (0) 
    // stop recursion at depth D-1 if NEE is true
    //else if (payload.depth == (cf.maxDepth /*- 1 - cf.next_event_est*/)) {
    //    //rtPrintf("is this ever called?");
    //    result += L_e;
    //    payload.radiance = result;
    //    payload.done = true;
    //}
    else {
        if (cf.next_event_est) {
            //if (attrib.objType == LIGHT) result = make_float3(.0f);
            result += L_d;
        }
        else {
            result += L_e;
        }
        payload.radiance = result * payload.throughput;
    }

    // Apply russian roulette to determine whether to terminate ray
    // or boost its contribution
    float q;
    if (cf.russian_roul) {
        q = 1.0f - fmin(fmax(fmax(payload.throughput.x, payload.throughput.y), payload.throughput.z), 1.0f);
        // pick a num from 0 to 1, if less than q, terminate ray
        // i.e. make throughput 0
        if (rnd(payload.seed) < q) {
            addon_throughput *= make_float3(.0f);
        }
        else {
            float thru_put_boost = (1.0f / (1.0f - q));
            addon_throughput *= thru_put_boost;
        }
    }
    payload.throughput *= addon_throughput;
    payload.origin = attrib.intersection;
    payload.dir = w_i;

    payload.depth++;
}

