#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"
#include "Config.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;
rtBuffer<QuadLight> qlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

rtBuffer<Config> config; // Config

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void closestHit()
{
    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float3 result = mv.ambient + mv.emission;

    // Calculate the direct illumination of point lights
    for (int i = 0; i < plights.size(); i++)
    {
        // Shoot a shadow to determin whether the object is in shadow
        float3 lightDir = normalize(plights[i].location - attrib.intersection);
        float lightDist = length(plights[i].location - attrib.intersection);
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;
        Ray shadowRay = make_Ray(attrib.intersection + lightDir * cf.epsilon, 
            lightDir, 1, cf.epsilon, lightDist);
        rtTrace(root, shadowRay, shadowPayload);

        // If not in shadow
        if (shadowPayload.isVisible)
        {
            float3 H = normalize(lightDir + attrib.wo);
            float att = dot(plights[i].attenuation, make_float3(1, lightDist, lightDist * lightDist));
            float3 I = mv.diffuse * fmaxf(dot(attrib.normal, lightDir), 0);
            I += mv.specular * pow(fmaxf(dot(attrib.normal, H), 0), mv.shininess);
            I *= plights[i].color / att;
            result += I;
        }
    }

    // Calculate the direct illumination of directional lights
    for (int i = 0; i < dlights.size(); i++)
    {
        // Shoot a shadow to determin whether the object is in shadow
        float3 lightDir = dlights[i].direction;
        float lightDist = RT_DEFAULT_MAX;
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;
        Ray shadowRay = make_Ray(attrib.intersection + lightDir * cf.epsilon, 
            lightDir, 1, cf.epsilon, lightDist);
        rtTrace(root, shadowRay, shadowPayload);

        // If not in shadow
        if (shadowPayload.isVisible)
        {
            float3 H = normalize(lightDir + attrib.wo);
            float3 I = mv.diffuse * fmaxf(dot(attrib.normal, lightDir), 0);
            I += mv.specular * pow(fmaxf(dot(attrib.normal, H), 0), mv.shininess);
            I *= dlights[i].color;
            result += I;
        }
    }

    // Another for-loop here to calculate contribution of quadLights
    // eg. for (int i = 0; i < qlights.size(); ++i) {}
    for (int i = 0; i < qlights.size(); ++i) {
        float3 f_brdf = mv.diffuse / M_PIf;// brdf function 
        float3 hitPt = attrib.intersection;
        float3 hitPtNormal = attrib.normal;

        float3 a = qlights[i].tri1->v1;
        float3 b = qlights[i].tri1->v2;
        float3 c = qlights[i].tri2->v2;
        float3 d = qlights[i].tri1->v3;


        float theta_1 = acosf(dot(normalize(a - hitPt), normalize(b - hitPt)));
        float theta_2 = acosf(dot(normalize(b - hitPt), normalize(c - hitPt)));
        float theta_3 = acosf(dot(normalize(c - hitPt), normalize(d - hitPt)));
        float theta_4 = acosf(dot(normalize(d - hitPt), normalize(a - hitPt)));

        float3 gamma_1 = normalize(cross((a - hitPt), (b - hitPt)));
        float3 gamma_2 = normalize(cross((b - hitPt), (c - hitPt)));
        float3 gamma_3 = normalize(cross((c - hitPt), (d - hitPt)));
        float3 gamma_4 = normalize(cross((d - hitPt), (a - hitPt)));

        float3 irradiance_vec = 0.5f * (theta_1 * gamma_1 + 
            theta_2 * gamma_2 + theta_3 * gamma_3 * theta_4 * gamma_4);

        float3 dir_radiance = f_brdf * qlights[i].color * dot(irradiance_vec, hitPtNormal);
        result += dir_radiance;
    }

    // Compute the final radiance
    payload.radiance = result * payload.throughput;

    // Calculate reflection
    if (length(mv.specular) > 0)
    {
        // Set origin and dir for tracing the reflection ray
        payload.origin = attrib.intersection;
        payload.dir = reflect(-attrib.wo, attrib.normal); // mirror reflection

        payload.depth++;
        payload.throughput *= mv.specular;
    }
    else
    {
        payload.done = true;
    }
}