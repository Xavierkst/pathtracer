#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

RT_PROGRAM void closestHit()
{
    // TODO: calculate the color using the Blinn-Phong reflection model

    //float3 result = make_float3(0, 1, 0);
    float3 result = attrib.ambient;
    result += dlights[0].light_color;
    for (int i = 0; i < plights.size(); i++) {
        result += plights[i].light_color;
    }
    payload.radiance = result;
}