#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"
#include "Config.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;
rtBuffer<QuadLight> qlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

rtBuffer<Config> config; // Config

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

rtDeclareVariable(uint, light_samples, , );
rtDeclareVariable(uint, light_stratify, , );
rtDeclareVariable(uint, next_event_est, , );
rtDeclareVariable(uint, sampling_method, , );

RT_PROGRAM void closestHit()
{
    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float3 result = mv.ambient + mv.emission;

    // Calculate the direct illumination of point lights
    for (int i = 0; i < plights.size(); i++)
    {
        // Shoot a shadow to determin whether the object is in shadow
        float3 lightDir = normalize(plights[i].location - attrib.intersection);
        float lightDist = length(plights[i].location - attrib.intersection);
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;
        Ray shadowRay = make_Ray(attrib.intersection + lightDir * cf.epsilon, 
            lightDir, 1, cf.epsilon, lightDist);
        rtTrace(root, shadowRay, shadowPayload);

        // If not in shadow
        if (shadowPayload.isVisible)
        {
            float3 H = normalize(lightDir + attrib.wo);
            float att = dot(plights[i].attenuation, make_float3(1, lightDist, lightDist * lightDist));
            float3 I = mv.diffuse * fmaxf(dot(attrib.normal, lightDir), 0);
            I += mv.specular * pow(fmaxf(dot(attrib.normal, H), 0), mv.shininess);
            I *= plights[i].color / att;
            result += I;
        }
    }

    // Calculate the direct illumination of directional lights
    for (int i = 0; i < dlights.size(); i++)
    {
        // Shoot a shadow to determin whether the object is in shadow
        float3 lightDir = dlights[i].direction;
        float lightDist = RT_DEFAULT_MAX;
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;
        Ray shadowRay = make_Ray(attrib.intersection + lightDir * cf.epsilon, 
            lightDir, 1, cf.epsilon, lightDist);
        rtTrace(root, shadowRay, shadowPayload);

        // If not in shadow
        if (shadowPayload.isVisible)
        {
            float3 H = normalize(lightDir + attrib.wo);
            float3 I = mv.diffuse * fmaxf(dot(attrib.normal, lightDir), 0);
            I += mv.specular * pow(fmaxf(dot(attrib.normal, H), 0), mv.shininess);
            I *= dlights[i].color;
            result += I;
        }
    }

    // Compute the final radiance
    payload.radiance = result * payload.throughput;

    // Calculate reflection
    if (length(mv.specular) > 0)
    {
        // Set origin and dir for tracing the reflection ray
        payload.origin = attrib.intersection;
        payload.dir = reflect(-attrib.wo, attrib.normal); // mirror reflection

        payload.depth++;
        payload.throughput *= mv.specular;
    }
    else
    {
        payload.done = true;
    }
}

RT_PROGRAM void analyticDirect() {

    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float3 result = mv.ambient + mv.emission;

    // for-loop here to calculate contribution of quadLights
    // if no light samples, do analytical direct:
    if (light_samples == 0) {
        for (int i = 0; i < qlights.size(); ++i) {
            float3 f_brdf = mv.diffuse / M_PIf;// brdf function 
            float3 hitPt = attrib.intersection;
            float3 hitPtNormal = attrib.normal;

            float3 a = qlights[i].tri1.v1;
            float3 b = qlights[i].tri1.v2;
            float3 c = qlights[i].tri2.v2;
            float3 d = qlights[i].tri1.v3;

            float3 points[] = { qlights[i].tri1.v1, qlights[i].tri1.v2, qlights[i].tri2.v2, qlights[i].tri1.v3 };

            float3 p1 = points[0]; float3 p2 = points[1]; float3 p3 = points[2]; float3 p4 = points[3];
            float theta_1 = acosf(dot(normalize(p1 - hitPt), normalize(p2 - hitPt)));
            float theta_2 = acosf(dot(normalize(p2 - hitPt), normalize(p3 - hitPt)));
            float theta_3 = acosf(dot(normalize(p3 - hitPt), normalize(p4 - hitPt)));
            float theta_4 = acosf(dot(normalize(p4 - hitPt), normalize(p1 - hitPt)));

            float3 gamma_1 = normalize(cross((p1 - hitPt), (p2 - hitPt)));
            float3 gamma_2 = normalize(cross((p2 - hitPt), (p3 - hitPt)));
            float3 gamma_3 = normalize(cross((p3 - hitPt), (p4 - hitPt)));
            float3 gamma_4 = normalize(cross((p4 - hitPt), (p1 - hitPt)));

            float3 irradiance_vec = 0.5f * ((theta_1 * gamma_1) +
                (theta_2 * gamma_2) + (theta_3 * gamma_3) + (theta_4 * gamma_4));

            float3 dir_radiance = f_brdf * qlights[i].color * dot(irradiance_vec, hitPtNormal);
            result += dir_radiance;
        }
    }

    payload.radiance = result;

    payload.done = true;
}

RT_PROGRAM void direct() {

    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float3 result = mv.ambient + mv.emission;

    if (attrib.objType == LIGHT) {
        payload.radiance = result;
        payload.depth = cf.maxDepth;
        payload.done = true;
        return;
    }

    for (int k = 0; k < qlights.size(); ++k) {
        float3 sampled_result = make_float3(.0f);
        // Compute direct lighting equation for w_i_k ray, for k = 1 to N*N
        float3 a = qlights[k].tri1.v1;
        float3 b = qlights[k].tri1.v2;
        float3 c = qlights[k].tri2.v3;
        float3 d = qlights[k].tri2.v2;

        float3 ac = c - a;
        float3 ab = b - a;
        float area = fabsf(length(cross(ab, ac)));
        int root_light_samples = (int)sqrtf(light_samples);
        // check if stratify or random sampling
        // double for loop here 
        for (int i = 0; i < root_light_samples; ++i) {
            for (int j = 0; j < root_light_samples; ++j) {
                // generate random float vals u1 and u2
                float u1 = rnd(payload.seed);
                float u2 = rnd(payload.seed);

                float3 sampled_light_pos;
                if (light_stratify) {
                    sampled_light_pos = a + ((j + u1) * (ab / (float)root_light_samples)) +
                        ((i + u2) * (ac / (float)root_light_samples));
                }
                else {
                    sampled_light_pos = a + u1 * ab + u2 * ac;
                }
                float3 shadow_ray_origin = attrib.intersection /*+ attrib.normal * cf.epsilon*/;
                float3 shadow_ray_dir = normalize(sampled_light_pos - shadow_ray_origin);
                float light_dist = length(sampled_light_pos - shadow_ray_origin);
                Ray shadow_ray = make_Ray(shadow_ray_origin, shadow_ray_dir, 1, cf.epsilon, light_dist - cf.epsilon);

                ShadowPayload shadow_payload;
                shadow_payload.isVisible = true;
                rtTrace(root, shadow_ray, shadow_payload);

                if (shadow_payload.isVisible) {
                    // rendering equation here: 
                    //float3 w_i = sampled_light_pos;
                    float3 f_brdf = (mv.diffuse / M_PIf) +
                        (mv.specular * ((mv.shininess + 2.0f) / (2.0f * M_PIf)) *
                            powf(fmaxf(dot(normalize(reflect(-attrib.wo, attrib.normal)), normalize(sampled_light_pos - shadow_ray_origin)), .0f), mv.shininess));

                    float3 x_prime = sampled_light_pos;
                    float3 x = shadow_ray_origin;
                    float3 n = attrib.normal;
                    float3 n_light = normalize(cross(ab, ac));
                    float R = length(x - x_prime);
                    // note: normal should point AWAY from the hitpoint, i.e. dot(n_light, x - x_prime) < 0
                    float G = (1.0f / powf(R, 2.0f)) * fmaxf(dot(n, normalize(x_prime - x)), .0f) *
                        (fmaxf(dot(n_light, normalize(x_prime - x)), .0f));

                    sampled_result += f_brdf * G; 
                }
            }
        }
        // we've taken samples k=1 to N of a single quad light, and we do it for all quad lights
        result += qlights[k].color * sampled_result * (area / (float)light_samples);
    }

    payload.radiance = result;
    payload.done = true;
}

RT_CALLABLE_PROGRAM void computePolarAngles(uint sampling_method, float& phi, float& theta, float zeta[3], float t, float shininess) {

    switch (sampling_method) {
    case HEMISPHERE_SAMPLING:
        //rtPrintf("hemisphere here\n");
        phi = 2.0f * M_PIf * zeta[2];
        theta = acosf(zeta[1]);
        break;
    case COSINE_SAMPLING:
        //rtPrintf("cosine here\n");
        phi = 2.0f * M_PIf * zeta[2];
        theta = acosf(sqrtf(zeta[1]));
        break;
    case BRDF_SAMPLING:
        //rtPrintf("brdf here\n");
        // phi remains the same for either specular or diffuse pdf
        phi = 2.0f * M_PIf * zeta[2];
        if (zeta[0] > t)
            theta = acosf(sqrtf(zeta[1])); // theta_diffuse
        else
            theta = acosf(powf(zeta[1], (1.0f / (shininess + 1.0f)))); // theta_specular
        break;
    default:
        break;
    }
}

RT_CALLABLE_PROGRAM float3 phongBRDF(float3 wi, float3 wo, float3 reflect_vec, 
    float shininess, float3 diffuse, float3 specular) {

    return (diffuse / M_PIf) + 
        (specular * ((shininess + 2.0f) / (2.0f * M_PIf)) * 
            powf(fmaxf(dot(reflect_vec, wi), .0f), shininess));
}
    
RT_CALLABLE_PROGRAM float3 ggxBRDF(float3 wi, float3 wo, float3 n, float roughness, float3 specular) {
    float wi_dot_n_dir = dot(wi, n);
    float wo_dot_n_dir = dot(wo, n);
    float alpha = roughness;
    float3 K_s = specular;
    float3 h = normalize(wi + wo); // half angle
    float theta_h = acosf(dot(h, n)); 
    float cos_theta_h_4 = powf(cosf(theta_h), 4.0f);
    float alpha_tan_theta_h_sq = (alpha * alpha) + powf(tanf(theta_h), 2.0f);
    // make sure denom not 0, else set D to 0
    // microfacet distribution function, D: 
    float D = (alpha * alpha) / (M_PIf * cos_theta_h_4 * powf(alpha_tan_theta_h_sq, 2.0f));

    // shadow-masking function, G:  
    float G_1_wi = (wi_dot_n_dir > .0f) ? 2.0f / (1.0f + sqrtf(1.0f + (alpha * alpha) * powf(tanf(acosf(dot(wi, n))), 2.0f))) : .0f;
    float G_1_wo = (wo_dot_n_dir > .0f) ? 2.0f / (1.0f + sqrtf(1.0f + (alpha * alpha) * powf(tanf(acosf(dot(wo, n))), 2.0f))) : .0f;
    float G = G_1_wi * G_1_wo;
    // fresnel function, F:
    float3 F = K_s + (1.0f - K_s) * powf(1.0f - dot(wi, h), 5.0f);
    return (F * G * D) / (4.0f * wi_dot_n_dir * wo_dot_n_dir);
}

RT_CALLABLE_PROGRAM float ggxPDF(float3 wi, float3 wo, float3 n, float3 half_angle_vec, float t_val, float roughness) {
    float alpha = roughness;
    float3 h = normalize(wi + wo); // half angle: 
    float theta_h = acosf(dot(half_angle_vec, n));
    float alpha_tan_theta_h_sq = (alpha * alpha) + powf(tanf(theta_h), 2.0f);
    float D = (alpha * alpha) / (M_PIf * powf(cosf(theta_h), 4.0f) * powf(alpha_tan_theta_h_sq, 2.0f));
    return ((1.0f - t_val) * fmaxf(dot(n, wi), .0f) / M_PIf) + ((t_val * D * dot(n, half_angle_vec)) / (4.0f * dot(half_angle_vec, wi)));
}

RT_CALLABLE_PROGRAM void swapValue(float& v1, float& v2) {
    float temp = v1;
    v1 = v2;
    v2 = temp;
}

RT_CALLABLE_PROGRAM void genCoordFrame(float3 w, float3& u, float3& v) {
    // incase a and w are closely aligned, swap a out for 
    // a diff arbitrary vector <1,0,0> instead of <0,1,0>
    float3 a = make_float3(.0f, 1.0f, .0f);
    // if (1.0f - fabsf(dot(a, w)) <= 1.0f) {
    if (1.0f - fabsf(dot(a, v)) <= 1.0f) {
        a = make_float3(1.0f, .0f, .0f);
    }
    u = normalize(cross(a, w)); 
    v = normalize(cross(w, u)); 
}

// Generates a new w_i direction:
// components u, v, w correspond to x, y and -wo
RT_CALLABLE_PROGRAM float3 sphericalDir(float theta, float phi, float3 u, float3 v, float3 w) {
    float3 sample_s = make_float3(cosf(phi) * 
        sinf(theta), sinf(phi) * sinf(theta), cosf(theta));

    return normalize((sample_s.x * u +
        sample_s.y * v + sample_s.z * w));
}

// As per PBR book on volume scattering, we create phase function to 
// vary each scatter direction within the volume. Simulates scattering behavior 
// by light rays
RT_CALLABLE_PROGRAM float phaseHG(float wi_dot_wo, float g) {
    float numerator = (1.0f - (g * g)); 
    float denom = powf((1.0f + powf(g, 2.0f) + (2.0f * g * wi_dot_wo)), (3.0f / 2.0f));

    return (1.0f / (4.0f * M_PIf)) * (numerator / denom);
}

// Compute the value for a given phase function
// and you don't necessarily have to sample a new dir w_i unless you want to. 
// You can just get the phase function value
RT_CALLABLE_PROGRAM float samplePhaseHG(float3 wo, float3& wi, float g, float rando_samples[]) {

    // find cos_theta val for phaseHG function
    float cos_theta = .0f;
    // if g == 0 
    if (fabsf(g) < 0.001f) {
        cos_theta = 1.0f - (2.0f * rando_samples[0]); // we'll default cos_theta to this value 
    }
    else { // for g > 0 
        cos_theta = (1.0f / (2.0f * g)) * 
            (1.0f + powf(g, 2.0f) - powf((1 - g * g) / (1 - g + (2.0f * g * rando_samples[0])), 2.0f));
    }
    // find sin_theta: 
    float sin_theta = sqrtf(fmaxf(0.f, 1.0f - (cos_theta * cos_theta)));
    // find sampling direction w_i (i.e the next scatter direction)
    float phi = 2.0f * M_PIf * rando_samples[1];
    float3 v1 = make_float3(.0f);
    float3 v2 = make_float3(.0f);

    genCoordFrame(wo, v1, v2);
    // compute new sampling dir w_i: 
    wi = sphericalDir(acosf(cos_theta), phi, v1, v2, wo);

    return phaseHG(-cos_theta, g);
}

RT_CALLABLE_PROGRAM float computeTransmittance(float sigma_t, float param_dist_t, float ray_length) {
    float tr = expf(-sigma_t * fminf(param_dist_t, RT_DEFAULT_MAX) * ray_length); 
    return tr;
}

RT_CALLABLE_PROGRAM float computeScatterDist(float zeta, float sigma_t) {
    // scatter distance as per siggraph course p. 35
    float scatter_dist = -(logf(1.0f - zeta)) / sigma_t;
    return scatter_dist;
}

RT_PROGRAM void pathTracer2()
{
    Config cf = config[0];
    float3 L_d = make_float3(.0f);
    float3 L_e = attrib.mv.emission;  

    // Terminate any indirect rays that intersect a light source
    if (payload.depth > 0 && attrib.objType == LIGHT) {
        payload.radiance = make_float3(.0f);
        payload.depth = cf.maxDepth;
        payload.done = true;
        return;
    }

    // Compute direct lighting
    for (int k = 0; k < qlights.size(); ++k) {
        int rt_light_samples = (int)sqrtf(light_samples);
        float3 a = qlights[k].tri1.v1;
        float3 b = qlights[k].tri1.v2;
        float3 c = qlights[k].tri1.v3;
        float3 d = qlights[k].tri2.v2;
        float3 ab = b - a;
        float3 ac = c - a;
        float area = fabsf(length(cross(ab, ac)));
        float3 sampled_radiance = make_float3(.0f);

        for (int i = 0; i < rt_light_samples; ++i) {
            for (int j = 0; j < rt_light_samples; ++j) {
                float u1 = rnd(payload.seed);
                float u2 = rnd(payload.seed);
                float3 light_sample_pos;
                if (light_stratify) {
                    // each subdivision cell of the light will be sampled
                    light_sample_pos = a + (j + u1) * (ab / (float) rt_light_samples) + (i + u2) * (ac / (float) rt_light_samples);
                }
                else { 
                    light_sample_pos = a + ab * u1 + ac * u2;
                }

                float3 light_sample_dir = normalize(light_sample_pos - attrib.intersection);
                float dist = length(light_sample_pos - attrib.intersection);
                float3 n_light = normalize(cross(ab, ac));
                // ray type 1 is shadow ray
                Ray shadow_ray = make_Ray(attrib.intersection, light_sample_dir, 1, cf.epsilon, dist - cf.epsilon);
                ShadowPayload shadow_payload;
                shadow_payload.isVisible = true; // assume point x not occluded b4 trace to light
                // Check if ray is occluded, if not, accumulate radiance frm light src
                rtTrace(root, shadow_ray, shadow_payload);
                // if not occluded, sample light radiance at that point (the V() term)
                if (shadow_payload.isVisible) {
                    // calculate BRDF, G(), V()
                    float3 reflect_dir = reflect(-attrib.wo, attrib.normal);
                    float G = (1.0f / (dist * dist))
                        * fmaxf(dot(attrib.normal, light_sample_dir), .0f)
                        * fmaxf(dot(n_light, light_sample_dir), .0f);
                    float3 f_brdf = phongBRDF(light_sample_dir, attrib.wo, reflect_dir, 
                        attrib.mv.shininess, attrib.mv.diffuse, attrib.mv.specular);

                    sampled_radiance += f_brdf * G;
                }
            }
        }
        L_d += qlights[k].color * (area / (float) light_samples) * sampled_radiance;
    }

    float zeta1 = rnd(payload.seed), zeta2(rnd(payload.seed));
    float theta = acosf(zeta1), phi = 2.0f * M_PIf * zeta2;
    // making an orthonormal basis for the hemisphere we're sampling 
    float3 s = make_float3(cosf(phi) * sinf(theta), sinf(phi) * sinf(theta), cosf(theta));
    // make the z-axis align with the hemisphere's normal
    float3 w = normalize(attrib.normal), a = make_float3(0, 1, 0);
    if (1.0f - fabsf(dot(a, w)) <= 1.0f) {
        // the vectors are too closely aligned, try another arbitrary a
        a = make_float3(1, 0, 0);
    }

    float3 u = normalize(cross(a, w)), v = normalize(cross(w, u));
    // generate w_i with the new orthonormal basis 
    float3 w_i = normalize(sphericalDir(theta, phi, u, v, w));
    float3 norm = normalize(attrib.normal);
    float3 incoming_ray_dir = normalize(-attrib.wo);
    float3 reflected_ray_dir = normalize(reflect(incoming_ray_dir, norm));
    float3 f_brdf = phongBRDF(w_i, normalize(attrib.wo), reflected_ray_dir, attrib.mv.shininess, attrib.mv.diffuse, attrib.mv.specular);
    float pdf = 1.0f / (2.0f * M_PIf);
    float3 attenuation = (f_brdf * fmaxf(dot(norm, w_i), .0f)) * (1.0f / pdf);
    
    // First ray cast should accumulate emission term from objects 
    if (payload.depth == 0) {
        payload.radiance += L_e * payload.throughput;
    }

    payload.radiance += L_d * payload.throughput;
    payload.throughput *= attenuation;
    payload.origin = attrib.intersection;
    payload.dir = w_i;
    payload.depth++;
}

RT_PROGRAM void pathTracer() {
    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float3 result = make_float3(.0f);
    float3 L_d = make_float3(.0f);
    float3 brdf_sampling_contribution = make_float3(.0f);
    float3 n = normalize(attrib.normal);
    float3 L_e = mv.emission;
    float3 K_s = mv.specular;
    float3 K_d = mv.diffuse;
    float3 r = normalize(reflect(-attrib.wo, attrib.normal));
    float theta = .0f;
    float phi = .0f;
    float exp_beta = 2.0f;
    float transmit = .0f;
    float3 w = make_float3(.0f);
    float3 w_i = make_float3(.0f);
    
    float3 addon_throughput = make_float3(1.0f);
    float3 sampled_result = make_float3(1.0f);
    float3 new_pos = make_float3(.0f);
    
    bool inMedium = false;;

    // variables for direct lighting
    int root_light_samples = (int)sqrtf(light_samples);
    float pdf_NEE = .0f;

    // When next event estimation is ON:
    // if an indir ray ever strikes light source (and it is NOT the first ray cast)
    // ray should be terminated
    if (cf.next_event_est && attrib.objType == LIGHT && payload.depth != 0) {
        payload.depth = cf.maxDepth;
        payload.done = true;
        payload.radiance = result;
        return;
    }

    // Add indirect lighting here:
    // generate randomize ray direction w_i
    float zeta_0 = rnd(payload.seed);
    float zeta_1 = rnd(payload.seed);
    float zeta_2 = rnd(payload.seed);
    float K_s_avg = (K_s.x + K_s.y + K_s.z) / 3.0f;
    float K_d_avg = (K_d.x + K_d.y + K_d.z) / 3.0f;
    float t = .0f;

    // make sure denom not 0, else check if phong or ggx
    // and set t accordingly
    if ((K_s_avg == .0f) && (K_d_avg == .0f)) {
        //rtPrintf("heeeere\n");
        // if denom is 0, t is 0 when brdf is modified phong, and 1 for ggx
        if (mv.brdf_type == MOD_PHONG)
            t = .0f;
        else if (mv.brdf_type == GGX)
            t = 1.0f;
    }
    else {
        // denom is not zero
        float k_val = K_s_avg / (K_s_avg + K_d_avg);
        if (mv.brdf_type == MOD_PHONG) {
            t = k_val;
        }
        else {
            t = fmaxf(.25f, k_val);
        }
    }
    
    float rand_vals[3] = { zeta_0, zeta_1, zeta_2 };
    computePolarAngles(sampling_method, phi, theta, rand_vals, t, mv.shininess);

    // we intersected the geometry, now we want to decide to cast either reflection or refraction ray
    // 1. Check material of surface before calling fresnel_schlick: 
    // if reflective, continue reflection as per normal
    // else if refractive: we have to make a choice whether ray will reflect or transmit into medium

    // Compute w_i with the refracted angle
    if (mv.matType == GLASS) {
        float3 inc_ray_dir = -attrib.wo;
        float3 norm = n;
        // we want to choose if we're going to reflect or refract this ray: 
        // if refract, use refract_vec below as the new w_i, else reflect just use 
        //  standard sampling technique (below)

        // we make w = n or -n
        // by checking if incoming ray is inside or outside the surface
        float n1 = 1.0f; float n2 = mv.ior;
        float eta = .0f;
        //float i_dot_n = dot(inc_ray_dir, norm);
        float i_dot_n = .0f;
        float reflect_probability = .0f;
        float3 chosen_ray_dir = make_float3(.0f);
        if (dot(inc_ray_dir, norm) >= .0f) { // incoming ray inside surface
            //norm = -norm;
            swapValue(n1, n2); 
            i_dot_n = dot(inc_ray_dir, norm); 
            inMedium = true;
        }
        else { // outside of surface
            // no change in normal or n1, n2
            i_dot_n = dot(inc_ray_dir, -norm);
            inMedium = false;
        }
        // check if refraction is possible, if yes, there's
        // a chance for refract, else, reflection only
        eta = n2 / n1;
        // check if either reflect/TIR, or refraction
        if (refract(chosen_ray_dir, inc_ray_dir, norm, eta)) {
            // reflection chance less than 100% if there is a refract ray
            reflect_probability = fresnel_schlick(i_dot_n);
            //rtPrintf("not not herere???\n");
        }
        else {
            //rtPrintf("abwout what herere???\n");
            reflect_probability = 1.0f; // completely reflected
        }
        // now choose between reflect or refract ray 
        float rand_valz = rnd(payload.seed);
        if (rand_valz < reflect_probability) { // reflection chosen
            w_i = normalize(reflect(inc_ray_dir, norm));
        }
        else { // refract
            w_i = normalize(chosen_ray_dir); 
        }
    }
    else if (mv.matType == VOLUMETRIC) {
        // when we intersect the homogeneous vol., we transmit inside and scatter 
        // check if currently inside or outside the volume
        float3 inc_ray_dir = -attrib.wo;
        float3 norm = n;
        // if outside medium, w_i remains the same direction
        // compute radiance on this surface 
    
        //rtPrintf("sigma_a %f sigma_s %f sigma_t %f\n", mv.sigma_a, mv.sigma_s, mv.sigma_t);
        // if inside medium, w_i will have new scatter direction
        if (dot(inc_ray_dir, norm) >= .0f) { // incoming ray inside surface
            w_i = payload.dir;
        }
        // else if outside medium, w_i will take the same scatter direction
        else {
            // get distance of ray segment
            float3 surface_pt = attrib.intersection;
            float3 curr_pt = attrib.prev_intersection;
            float ray_distance = length(surface_pt - curr_pt);
            // compute parametric dist t that spans the ray segment: -- sampling method of exp distrib, pg 893
            //float param_dist_t = computeScatterDist(zeta_0, mv.sigma_t);
            float param_dist_t = 0.08f;

            //rtPrintf("param_dist_t %f\n", param_dist_t);
            // compute transmittance and add it to throughput 
            transmit = computeTransmittance(mv.sigma_t, param_dist_t, ray_distance);
            
            // what about phase function value? need to add it on to throughput? 
            // phase fn takes place of weight
            float3 place_holder_vec = make_float3(.0f);
            float weight = samplePhaseHG(inc_ray_dir, place_holder_vec, mv.g, rand_vals);
            
            addon_throughput *= transmit * weight; // do we need to add weight to throughput?

            // check if sampled distance overshoots ray segment distance
            // if not, we sample vol. for anoth scatt dir w_i
            if (param_dist_t < ray_distance) {
                // get scatter direction -- 
                // we try single scattering first, so just march along the ray
                w_i = payload.dir;
                new_pos = curr_pt + (w_i * param_dist_t);
                rtPrintf("curr pt: %f %f %f \n", curr_pt.x, curr_pt.y, curr_pt.z);
                //rtPrintf("new_pos: %f %f %f \n", new_pos.x, new_pos.y, new_pos.z);
            }
            else {
                rtPrintf("or here here\n");
                w_i = payload.dir;
                new_pos = surface_pt;
            }
        }
    }
    else {  // if the material is not glass, we sample normally
            // get new spherical ray dir -- choose a sampling method
        if (mv.brdf_type == MOD_PHONG) { // Phong 
            // generate coordinate frame at the intersect point
            // if specular brdf chosen, center sample_s at reflection vector r
            w = ((zeta_0 <= t) && (sampling_method == BRDF_SAMPLING)) ? r : n;
            float3 u = make_float3(.0f);
            float3 v = make_float3(.0f);
            // qn: why rotate s wrt the z-axis? and not the y-axis?
            genCoordFrame(w, u, v);
            w_i = sphericalDir(theta, phi, u, v, w);
        }
        else { // GGX sampling
            float theta_h_sample = atanf((mv.roughness * sqrtf(zeta_1)) / sqrtf(1.0f - zeta_1));
            float phi_h_sample = 2.0f * M_PIf * zeta_2;
            float3 w_h = n;
            float3 u_h = make_float3(.0f);
            float3 v_h = make_float3(.0f);
            // generate coordinate frame
            genCoordFrame(w_h, u_h, v_h);
            // compute new w_i direction
            w_i = normalize(reflect(-attrib.wo, 
                sphericalDir(theta_h_sample, phi_h_sample, u_h, v_h, w_h)));
        }
    }

    // Calculating dir light contribution from NEE
    if (cf.next_event_est) {
        // Add direct lighting here:
        for (int k = 0; k < qlights.size(); ++k) {
            sampled_result = make_float3(.0f);
            float pdf_lights_k = .0f;
            float pdf_brdf = .0f;
            float3 w_i_dir = make_float3(.0f);
            float3 h = make_float3(.0f);
            float D = .0f;
            // Compute direct lighting equation for w_i_k ray, for k = 1 to N*N
            float3 a = qlights[k].tri1.v1;
            float3 b = qlights[k].tri1.v2;
            float3 c = qlights[k].tri2.v3;
            float3 d = qlights[k].tri2.v2;

            float3 ac = c - a;
            float3 ab = b - a;
            float area = length(cross(ab, ac));
            int root_light_samples = (int)sqrtf(light_samples);
            // check if stratify or random sampling
            // double for loop here 
            for (int i = 0; i < root_light_samples; ++i) {
                for (int j = 0; j < root_light_samples; ++j) {
                    // generate random float vals u1 and u2
                    float u1 = rnd(payload.seed);
                    float u2 = rnd(payload.seed);

                    float3 sampled_light_pos;
                    if (light_stratify) {
                        sampled_light_pos = a + ((j + u1) * (ab / (float)root_light_samples)) +
                            ((i + u2) * (ac / (float)root_light_samples));
                    }
                    else {
                        sampled_light_pos = a + u1 * ab + u2 * ac;
                    }

                    float3 shadow_ray_origin = attrib.intersection /*+ attrib.normal * cf.epsilon*/;
                    float3 shadow_ray_dir = normalize(sampled_light_pos - shadow_ray_origin);
                    float light_dist = length(sampled_light_pos - shadow_ray_origin);
                    Ray shadow_ray = make_Ray(shadow_ray_origin, shadow_ray_dir, 1, cf.epsilon, light_dist - cf.epsilon);

                    ShadowPayload shadow_payload;
                    shadow_payload.isVisible = true;
                    rtTrace(root, shadow_ray, shadow_payload);

                    if (shadow_payload.isVisible) {
                        //float D = .0f;

                        // rendering equation here: 
                        //float3 w_i = sampled_light_pos;
                        w_i_dir = normalize(sampled_light_pos - shadow_ray_origin);

                        float3 f_brdf = make_float3(.0f);
                        if (mv.brdf_type == MOD_PHONG) {
                            f_brdf = phongBRDF(w_i_dir, attrib.wo, r, mv.shininess, mv.diffuse, mv.specular);
                        }
                        else { // GGX 
                            float wi_dot_n_dir = dot(w_i_dir, n);
                            float wo_dot_n_dir = dot(attrib.wo, n);
                            if (wi_dot_n_dir > .0f && wo_dot_n_dir > .0f) {
                                // compute ggx material BRDF 
                                float3 f_brdf_GGX = ggxBRDF(w_i_dir, attrib.wo, n, mv.roughness, K_s);
                                f_brdf = (K_d / M_PIf) + f_brdf_GGX;
                            }
                        }
                        float3 x_prime = sampled_light_pos;
                        float3 x = shadow_ray_origin;
                        float3 n = attrib.normal;
                        float3 n_light = normalize(cross(ab, ac));

                        float R = length(x - x_prime);

                        // note: normal should point AWAY from the hitpoint, i.e. dot(n_light, x - x_prime) < 0
                        float G = (1.0f / powf(R, 2.0f)) * fmaxf(dot(n, normalize(x_prime - x)), .0f) *
                            (fmaxf(dot(n_light, normalize(x_prime - x)), .0f));

                        sampled_result += f_brdf * G;
                        //pdf_lights_k += (powf(R, 2.0f) / (area * fabsf(dot(n, w_i_dir)))); 
                    }
                }
            }
            // calculate weight Wi for this given w_i generated
            pdf_NEE = pdf_lights_k; /** (1.0f / (float) qlights.size())*/

            // calculate pdf_brdf 
            pdf_brdf = ((1.0f - t) * fmaxf(dot(n, w_i_dir), .0f) / M_PIf) + ((t * D * dot(n, h)) / (4.0f * dot(h, w_i_dir)));
         
            // calc power heuristic: 
            float pdf_denom_sum = powf(pdf_NEE, exp_beta) + powf(pdf_brdf, exp_beta);
            float pdf_numerator = powf(pdf_NEE, exp_beta);
            float weight_i = pdf_numerator / pdf_denom_sum;
        
            if (cf.next_event_est == MIS) {
                //rtPrintf("here");
                L_d += qlights[k].color * sampled_result * (area / (float)light_samples) * (1.0f / pdf_NEE) * weight_i;
            }
            else {
                // divide brdf by the pdf here
                L_d += qlights[k].color * sampled_result * (area / (float)light_samples);
            }
        }
    }
    // calculate the summation of all pdfs here (i.e. pdf_nee + pdf_brdf)
    // using h, w_i, 

    // the BRDF 
    float3 f_brdf = make_float3(1.0f);
    float pdf = 1.0f;
    float phase_fn_value = 1.0f;

    //sampling_method = 5;
    switch (sampling_method) {
    case HEMISPHERE_SAMPLING:
        //rtPrintf("hemisphere here");
        f_brdf = (mv.diffuse / M_PIf) +
            (mv.specular * ((mv.shininess + 2.0f) / (2.0f * M_PIf)) *
                powf(fmaxf(dot(r, w_i), .0f), mv.shininess));
        pdf = 1.0f / (2.0f * M_PIf);
        addon_throughput = (f_brdf * fmaxf(dot(n, w_i), .0f) * (1.0f / pdf));
        break;
    case COSINE_SAMPLING:
        //rtPrintf("cosine here");
        f_brdf = (mv.diffuse / M_PIf) +
            (mv.specular * ((mv.shininess + 2.0f) / (2.0f * M_PIf)) *
                powf(fmaxf(dot(r, w_i), .0f), mv.shininess));
        pdf = fmaxf(dot(n, w_i), .0f) / (M_PIf);
        addon_throughput = (f_brdf)*fmaxf(dot(n, w_i), .0f) * (1.0f / pdf);
        break;
    case BRDF_SAMPLING:
        float pdf_NEE_brdf = .0f;
        float pdf_lights_k = .0f;
            for (int k = 0; k < qlights.size(); ++k) {
                    //float3 sampled_result = make_float3(.0f);
                    //float pdf_brdf = .0f;
                    // Compute direct lighting equation for w_i_k ray, for k = 1 to N*N
                    float3 a = qlights[k].tri1.v1;
                    float3 b = qlights[k].tri1.v2;
                    float3 c = qlights[k].tri2.v3;
                    float3 d = qlights[k].tri2.v2;

                    float3 ac = c - a;
                    float3 ab = b - a;
                    float area = length(cross(ab, ac));

                    float3 shadow_ray_origin = attrib.intersection /*+ attrib.normal * cf.epsilon*/;
                    // trace the ray and see if it hits a light source
                    Ray shadow_ray = make_Ray(shadow_ray_origin, w_i, 1, cf.epsilon, RT_DEFAULT_MAX);
                    //float light_dist = length(sampled_light_pos - shadow_ray_origin);
                    ShadowPayload shadow_payload;
                    shadow_payload.isVisible = true;
                    rtTrace(root, shadow_ray, shadow_payload);

                    if (!shadow_payload.isVisible && shadow_payload.objType == LIGHT) {
                        float3 x_prime = shadow_payload.intersectPt;
                        float3 x = shadow_ray_origin;
                        float3 n_light = normalize(cross(ab, ac));
                        float R = length(x - x_prime);

                        pdf_lights_k += (powf(R, 2.0f) / (area * fabsf(dot(n, w_i))));
                    }
            }
        // check the material whether to use mod-phong or GGX brdf
        if (mv.brdf_type == MOD_PHONG) {
            f_brdf = phongBRDF(w_i, attrib.wo, r, mv.shininess, K_d, K_s);

            pdf = ((1.0f - t) * (dot(n, w_i) / M_PIf)) +
                t * ((mv.shininess + 1.0f) / (2.0f * M_PIf)) *
                powf(fmaxf(dot(r, w_i), .0f), mv.shininess);
        }
        else { // Default: use GGX BRDF
            float wi_dot_n = dot(w_i, n);
            float wo_dot_n = dot(attrib.wo, n);
            
            if (wi_dot_n > .0f && wo_dot_n > .0f) {
                float3 h = normalize(w_i + attrib.wo); // half angle: 
                float3 f_brdf_GGX = ggxBRDF(w_i, attrib.wo, n, mv.roughness, K_s);
                f_brdf = (K_d / M_PIf) + f_brdf_GGX;
                pdf = ggxPDF(w_i, attrib.wo, n, h, t, mv.roughness);
            }
            else f_brdf = make_float3(.0f); // assume f zero otherwise
            pdf_NEE_brdf = (1.0f / qlights.size()) * pdf_lights_k;
        }

        float pdf_denom_sum = powf(pdf_NEE_brdf, exp_beta) + powf(pdf, exp_beta);
        float pdf_numerator = powf(pdf, exp_beta);
        float weight_i_brdf = pdf_numerator / pdf_denom_sum;

        if (cf.next_event_est == MIS) {
            //mv.emission;
            float3 brdf_sampling_contribution = L_e* (f_brdf * fmaxf(dot(n, w_i), .0f) * (1.0f / pdf) * weight_i_brdf);
            addon_throughput = (f_brdf * fmaxf(dot(n, w_i), .0f) * (1.0f / pdf));
        }
        else {
            addon_throughput = (f_brdf * fmaxf(dot(n, w_i), .0f) * (1.0f / pdf));
        }
        break;
        default:
    }
    
    // Check if its first intersected surface
    if (cf.next_event_est && (payload.depth == 0)) {
        // use direct lighting and emission contribution from object for 1st bounce
        result += L_e + L_d; 
    }
    // on the last bounce, we return only emission term
    else {
        if (cf.next_event_est == ON || cf.next_event_est == MIS) {
            // only use direct lighting contribution if NEE
            result += L_d; 
        }
        //else if (cf.next_event_est == MIS) {
        //    result += brdf_sampling_contribution;
        //}
        else {
            // If no NEE, use only emission contribution from object
            result += L_e;
        }
    }
    payload.radiance = result * payload.throughput;
    
    // Apply russian roulette to determine whether to terminate ray
    // or boost its contribution
    float q = .0f;
    if (cf.russian_roul) {
        q = 1.0f - fmin(fmax(fmax(payload.throughput.x, payload.throughput.y), payload.throughput.z), 1.0f);
        // pick a num from 0 to 1, if less than q, terminate ray
        if (rnd(payload.seed) < q) {
            addon_throughput *= make_float3(.0f);
        }
        else {
            float thru_put_boost = (1.0f / (1.0f - q));
            addon_throughput *= thru_put_boost;
        }
    }
    
    if (mv.matType == GLASS) { // we don't diminish throughput of glass?
        payload.throughput *= make_float3(1.0f);
        //payload.throughput *= addon_throughput;
    }
    else {
        payload.throughput *= addon_throughput;
    }

    if (mv.matType == VOLUMETRIC) {
        //rtPrintf("%f\n", t_dist);
        payload.origin = new_pos;
        payload.dir = w_i;
    }
    else {
        //rtPrintf("hereref\n");
        if (mv.matType == GLASS) {
            payload.origin = new_pos;
        }
        else
            payload.origin = attrib.intersection;

        payload.dir = w_i;
    }

    if (mv.matType != GLASS || mv.matType != VOLUMETRIC) {
        //rtPrintf("not here here\n");
        payload.depth++;
    }
}
