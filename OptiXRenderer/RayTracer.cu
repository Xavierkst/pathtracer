#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"
#include "Config.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;
rtBuffer<QuadLight> qlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

rtBuffer<Config> config; // Config

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

rtDeclareVariable(uint, light_samples, , );
rtDeclareVariable(uint, light_stratify, , );
rtDeclareVariable(uint, next_event_est, , );

RT_PROGRAM void closestHit()
{
    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float3 result = mv.ambient + mv.emission;

    // Calculate the direct illumination of point lights
    for (int i = 0; i < plights.size(); i++)
    {
        // Shoot a shadow to determin whether the object is in shadow
        float3 lightDir = normalize(plights[i].location - attrib.intersection);
        float lightDist = length(plights[i].location - attrib.intersection);
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;
        Ray shadowRay = make_Ray(attrib.intersection + lightDir * cf.epsilon, 
            lightDir, 1, cf.epsilon, lightDist);
        rtTrace(root, shadowRay, shadowPayload);

        // If not in shadow
        if (shadowPayload.isVisible)
        {
            float3 H = normalize(lightDir + attrib.wo);
            float att = dot(plights[i].attenuation, make_float3(1, lightDist, lightDist * lightDist));
            float3 I = mv.diffuse * fmaxf(dot(attrib.normal, lightDir), 0);
            I += mv.specular * pow(fmaxf(dot(attrib.normal, H), 0), mv.shininess);
            I *= plights[i].color / att;
            result += I;
        }
    }

    // Calculate the direct illumination of directional lights
    for (int i = 0; i < dlights.size(); i++)
    {
        // Shoot a shadow to determin whether the object is in shadow
        float3 lightDir = dlights[i].direction;
        float lightDist = RT_DEFAULT_MAX;
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;
        Ray shadowRay = make_Ray(attrib.intersection + lightDir * cf.epsilon, 
            lightDir, 1, cf.epsilon, lightDist);
        rtTrace(root, shadowRay, shadowPayload);

        // If not in shadow
        if (shadowPayload.isVisible)
        {
            float3 H = normalize(lightDir + attrib.wo);
            float3 I = mv.diffuse * fmaxf(dot(attrib.normal, lightDir), 0);
            I += mv.specular * pow(fmaxf(dot(attrib.normal, H), 0), mv.shininess);
            I *= dlights[i].color;
            result += I;
        }
    }

    // Compute the final radiance
    payload.radiance = result * payload.throughput;

    // Calculate reflection
    if (length(mv.specular) > 0)
    {
        // Set origin and dir for tracing the reflection ray
        payload.origin = attrib.intersection;
        payload.dir = reflect(-attrib.wo, attrib.normal); // mirror reflection

        payload.depth++;
        payload.throughput *= mv.specular;
    }
    else
    {
        payload.done = true;
    }
}

RT_PROGRAM void analyticDirect() {

    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float3 result = mv.ambient + mv.emission;

    // for-loop here to calculate contribution of quadLights
    // if no light samples, do analytical direct:
    if (light_samples == 0) {
        for (int i = 0; i < qlights.size(); ++i) {
            float3 f_brdf = mv.diffuse / M_PIf;// brdf function 
            float3 hitPt = attrib.intersection;
            float3 hitPtNormal = attrib.normal;

            float3 a = qlights[i].tri1.v1;
            float3 b = qlights[i].tri1.v2;
            float3 c = qlights[i].tri2.v2;
            float3 d = qlights[i].tri1.v3;

            float3 points[] = { qlights[i].tri1.v1, qlights[i].tri1.v2, qlights[i].tri2.v2, qlights[i].tri1.v3 };

            float3 p1 = points[0]; float3 p2 = points[1]; float3 p3 = points[2]; float3 p4 = points[3];
            float theta_1 = acosf(dot(normalize(p1 - hitPt), normalize(p2 - hitPt)));
            float theta_2 = acosf(dot(normalize(p2 - hitPt), normalize(p3 - hitPt)));
            float theta_3 = acosf(dot(normalize(p3 - hitPt), normalize(p4 - hitPt)));
            float theta_4 = acosf(dot(normalize(p4 - hitPt), normalize(p1 - hitPt)));

            float3 gamma_1 = normalize(cross((p1 - hitPt), (p2 - hitPt)));
            float3 gamma_2 = normalize(cross((p2 - hitPt), (p3 - hitPt)));
            float3 gamma_3 = normalize(cross((p3 - hitPt), (p4 - hitPt)));
            float3 gamma_4 = normalize(cross((p4 - hitPt), (p1 - hitPt)));

            float3 irradiance_vec = 0.5f * ((theta_1 * gamma_1) +
                (theta_2 * gamma_2) + (theta_3 * gamma_3) + (theta_4 * gamma_4));

            float3 dir_radiance = f_brdf * qlights[i].color * dot(irradiance_vec, hitPtNormal);
            result += dir_radiance;
        }
    }

    payload.radiance = result;

    payload.done = true;
}


RT_PROGRAM void direct() {

    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float3 result = mv.ambient + mv.emission;

    for (int k = 0; k < qlights.size(); ++k) {
        float3 sampled_result = make_float3(.0f);
        // Compute direct lighting equation for w_i_k ray, for k = 1 to N*N
        float3 a = qlights[k].tri1.v1;
        float3 b = qlights[k].tri1.v2;
        float3 c = qlights[k].tri2.v3;
        float3 d = qlights[k].tri2.v2;

        float3 ac = c - a;
        float3 ab = b - a;
        float area = length(cross(ab, ac));
        int root_light_samples = (int)sqrtf(light_samples);
        // check if stratify or random sampling
        // double for loop here 
        for (int i = 0; i < root_light_samples; ++i) {
            for (int j = 0; j < root_light_samples; ++j) {
                // generate random float vals u1 and u2
                float u1 = rnd(payload.seed);
                float u2 = rnd(payload.seed);

                float3 sampled_light_pos;
                if (light_stratify) {
                    sampled_light_pos = a + ((j + u1) * (ab / (float)root_light_samples)) +
                        ((i + u2) * (ac / (float)root_light_samples));
                }
                else {
                    sampled_light_pos = a + u1 * ab + u2 * ac;
                }
                float3 shadow_ray_origin = attrib.intersection /*+ attrib.normal * cf.epsilon*/;
                float3 shadow_ray_dir = normalize(sampled_light_pos - shadow_ray_origin);
                float light_dist = length(sampled_light_pos - shadow_ray_origin);
                Ray shadow_ray = make_Ray(shadow_ray_origin, shadow_ray_dir, 1, cf.epsilon, light_dist - cf.epsilon);

                ShadowPayload shadow_payload;
                shadow_payload.isVisible = true;
                rtTrace(root, shadow_ray, shadow_payload);

                if (shadow_payload.isVisible) {
                    // rendering equation here: 
                    //float3 w_i = sampled_light_pos;
                    float3 f_brdf = (mv.diffuse / M_PIf) +
                        (mv.specular * ((mv.shininess + 2.0f) / (2.0f * M_PIf)) *
                            powf(fmaxf(dot(normalize(reflect(-attrib.wo, attrib.normal)), normalize(sampled_light_pos - shadow_ray_origin)), .0f), mv.shininess));

                    float3 x_prime = sampled_light_pos;
                    float3 x = shadow_ray_origin;
                    float3 n = attrib.normal;
                    //float3 n_light = normalize(qlights[k].tri1.normal);
                    float3 n_light = normalize(cross(ab, ac));
                    //n_light = dot(n_light, normalize(x_prime - x)) > .0f ? n_light : -n_light;

                    float R = length(x - x_prime);

                    // note: normal should point AWAY from the hitpoint, i.e. dot(n_light, x - x_prime) < 0
                    float G = (1.0f / powf(R, 2.0f)) * fmaxf(dot(n, normalize(x_prime - x)), .0f) *
                        (fmaxf(dot(n_light, normalize(x_prime - x)), .0f));

                    sampled_result += f_brdf * G;
                }
            }
        }
        result += qlights[k].color * sampled_result * (area / (float)light_samples);
    }
    //rtPrintf("throughput val: %f \n", payload.throughput);
    payload.radiance = result;

    payload.done = true;
}

RT_PROGRAM void pathTracer() {

    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float3 L_e = mv.emission;
    float3 result = make_float3(.0f);
    float3 L_d = make_float3(.0f);
    
    // When next event estimation is ON:
    // if an indir ray ever strikes light source (and it is NOT the first ray cast)
    // ray should be terminated
    if (cf.next_event_est && attrib.objType == LIGHT && payload.depth != 0) {
        payload.depth = cf.maxDepth;
        payload.done = true;
        payload.radiance = result;
        return;
    }
    if (cf.next_event_est) {

        // Add direct lighting here:
        for (int k = 0; k < qlights.size(); ++k) {
            float3 sampled_result = make_float3(.0f);
            // Compute direct lighting equation for w_i_k ray, for k = 1 to N*N
            float3 a = qlights[k].tri1.v1;
            float3 b = qlights[k].tri1.v2;
            float3 c = qlights[k].tri2.v3;
            float3 d = qlights[k].tri2.v2;

            float3 ac = c - a;
            float3 ab = b - a;
            float area = length(cross(ab, ac));
            int root_light_samples = (int)sqrtf(light_samples);
            // check if stratify or random sampling
            // double for loop here 
            for (int i = 0; i < root_light_samples; ++i) {
                for (int j = 0; j < root_light_samples; ++j) {
                    // generate random float vals u1 and u2
                    float u1 = rnd(payload.seed);
                    float u2 = rnd(payload.seed);

                    float3 sampled_light_pos;
                    if (light_stratify) {
                        sampled_light_pos = a + ((j + u1) * (ab / (float)root_light_samples)) +
                            ((i + u2) * (ac / (float)root_light_samples));
                    }
                    else {
                        sampled_light_pos = a + u1 * ab + u2 * ac;
                    }
                    float3 shadow_ray_origin = attrib.intersection /*+ attrib.normal * cf.epsilon*/;
                    float3 shadow_ray_dir = normalize(sampled_light_pos - shadow_ray_origin);
                    float light_dist = length(sampled_light_pos - shadow_ray_origin);
                    Ray shadow_ray = make_Ray(shadow_ray_origin, shadow_ray_dir, 1, cf.epsilon, light_dist - cf.epsilon);

                    ShadowPayload shadow_payload;
                    shadow_payload.isVisible = true;
                    rtTrace(root, shadow_ray, shadow_payload);

                    if (shadow_payload.isVisible) {
                        // rendering equation here: 
                        //float3 w_i = sampled_light_pos;
                        float3 f_brdf = (mv.diffuse / M_PIf) +
                            (mv.specular * ((mv.shininess + 2.0f) / (2.0f * M_PIf)) *
                                powf(fmaxf(dot(normalize(reflect(-attrib.wo, attrib.normal)), normalize(sampled_light_pos - shadow_ray_origin)), .0f), mv.shininess));

                        float3 x_prime = sampled_light_pos;
                        float3 x = shadow_ray_origin;
                        float3 n = attrib.normal;
                        float3 n_light = normalize(cross(ab, ac));

                        float R = length(x - x_prime);

                        // note: normal should point AWAY from the hitpoint, i.e. dot(n_light, x - x_prime) < 0
                        float G = (1.0f / powf(R, 2.0f)) * fmaxf(dot(n, normalize(x_prime - x)), .0f) *
                            (fmaxf(dot(n_light, normalize(x_prime - x)), .0f));

                        sampled_result += f_brdf * G;
                    }
                }
            }
            L_d += qlights[k].color * sampled_result * (area / (float)light_samples);
        }
    }

    // Add indirect lighting here:
    // generate randomize ray direction w_i
    float zeta_1 = rnd(payload.seed); 
    float zeta_2 = rnd(payload.seed);
    float theta = acosf(zeta_1); 
    float phi = 2.0f * M_PIf * zeta_2;

    // qn: why rotate s wrt the z-axis? and not the y-axis?
    float3 sample_s = make_float3(cosf(phi) * sinf(theta), sinf(phi) * sinf(theta), cosf(theta));
    
    // generate coordinate frame at the intersect point
    float3 n = attrib.normal;
    float3 w = normalize(n);
    float3 a = make_float3(.0f, 1.0f, .0f);
    // incase a and w are closely aligned, swap a out for 
    // a diff arbitrary vector <1,0,0> instead of <0,1,0>
    if (1.0f - fabsf(dot(a, w)) <= 1.0f) {
        a = make_float3(1.0f, .0f, .0f);
    }

    float3 u = normalize(cross(a, w));
    float3 v = normalize(cross(w, u)); // i dont think need to normalize

    // find randomized new ray dir
    float3 w_i = (sample_s.x * u + sample_s.y * v + sample_s.z * w);   

    // the BRDF 
    float3 f_brdf = (mv.diffuse / M_PIf) +
        (mv.specular * ((mv.shininess + 2.0f) / (2.0f * M_PIf)) *
            powf(fmaxf(dot(normalize(reflect(-attrib.wo, attrib.normal)), 
                w_i), .0f), mv.shininess));

    float3 addon_throughput = 2.0f * M_PIf * f_brdf * fmaxf(dot(n, w_i), .0f);

    // Check if its first intersected surface
    if (cf.next_event_est && (payload.depth == 0)) {
        result += L_e;
        payload.radiance = (result + L_d) * payload.throughput;
    }
    // on the last bounce, we return only emission term
    // NEE true (1) or false (0) 
    // stop recursion at depth D-1 if NEE is true
    //else if (payload.depth == (cf.maxDepth /*- 1 - cf.next_event_est*/)) {
    //    //rtPrintf("is this ever called?");
    //    result += L_e;
    //    payload.radiance = result;
    //    payload.done = true;
    //}
    else {
        if (cf.next_event_est) {
            //if (attrib.objType == LIGHT) result = make_float3(.0f);
            result += L_d;
        }
        else {
            result += L_e;
        }
        payload.radiance = result * payload.throughput;
    }

    float q;
    if (cf.russian_roul) {
        q = 1.0f - fmin(fmax(fmax(payload.throughput.x, payload.throughput.y), payload.throughput.z), 1.0f);
        // pick a num from 0 to 1, if less than q, terminate ray
        // i.e. make throughput 0
        if (rnd(payload.seed) < q) {
            addon_throughput *= make_float3(.0f);
        }
        else {
            float thru_put_boost = (1.0f / (1.0f - q));
            addon_throughput *= thru_put_boost;
        }
    }
    payload.throughput *= addon_throughput;
    payload.origin = attrib.intersection;
    payload.dir = w_i;

    payload.depth++;
}

