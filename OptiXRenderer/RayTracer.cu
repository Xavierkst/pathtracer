#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>
#include <optixu/optixu_math_namespace.h>
#include "random.h"

#include "Payloads.h"
#include "Geometries.h"
#include "Light.h"
#include "Config.h"

using namespace optix;

// Declare light buffers
rtBuffer<PointLight> plights;
rtBuffer<DirectionalLight> dlights;
rtBuffer<QuadLight> qlights;

// Declare variables
rtDeclareVariable(Payload, payload, rtPayload, );
rtDeclareVariable(rtObject, root, , );

rtBuffer<Config> config; // Config

// Declare attibutes 
rtDeclareVariable(Attributes, attrib, attribute attrib, );

rtDeclareVariable(uint, light_samples, , );
rtDeclareVariable(uint, light_stratify, , );

RT_PROGRAM void closestHit()
{
    MaterialValue mv = attrib.mv;
    Config cf = config[0];

    float3 result = mv.ambient + mv.emission;

    // Calculate the direct illumination of point lights
    for (int i = 0; i < plights.size(); i++)
    {
        // Shoot a shadow to determin whether the object is in shadow
        float3 lightDir = normalize(plights[i].location - attrib.intersection);
        float lightDist = length(plights[i].location - attrib.intersection);
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;
        Ray shadowRay = make_Ray(attrib.intersection + lightDir * cf.epsilon, 
            lightDir, 1, cf.epsilon, lightDist);
        rtTrace(root, shadowRay, shadowPayload);

        // If not in shadow
        if (shadowPayload.isVisible)
        {
            float3 H = normalize(lightDir + attrib.wo);
            float att = dot(plights[i].attenuation, make_float3(1, lightDist, lightDist * lightDist));
            float3 I = mv.diffuse * fmaxf(dot(attrib.normal, lightDir), 0);
            I += mv.specular * pow(fmaxf(dot(attrib.normal, H), 0), mv.shininess);
            I *= plights[i].color / att;
            result += I;
        }
    }

    // Calculate the direct illumination of directional lights
    for (int i = 0; i < dlights.size(); i++)
    {
        // Shoot a shadow to determin whether the object is in shadow
        float3 lightDir = dlights[i].direction;
        float lightDist = RT_DEFAULT_MAX;
        ShadowPayload shadowPayload;
        shadowPayload.isVisible = true;
        Ray shadowRay = make_Ray(attrib.intersection + lightDir * cf.epsilon, 
            lightDir, 1, cf.epsilon, lightDist);
        rtTrace(root, shadowRay, shadowPayload);

        // If not in shadow
        if (shadowPayload.isVisible)
        {
            float3 H = normalize(lightDir + attrib.wo);
            float3 I = mv.diffuse * fmaxf(dot(attrib.normal, lightDir), 0);
            I += mv.specular * pow(fmaxf(dot(attrib.normal, H), 0), mv.shininess);
            I *= dlights[i].color;
            result += I;
        }
    }

    // Another for-loop here to calculate contribution of quadLights
     //eg. for (int i = 0; i < qlights.size(); ++i) {}
    // if no light samples, do analytical direct:
    if (light_samples == 0) {
        for (int i = 0; i < qlights.size(); ++i) {
            float3 f_brdf = mv.diffuse / M_PIf;// brdf function 
            float3 hitPt = attrib.intersection;
            float3 hitPtNormal = attrib.normal;

            float3 a = qlights[i].tri1.v1;
            float3 b = qlights[i].tri1.v2;
            float3 c = qlights[i].tri2.v2;
            float3 d = qlights[i].tri1.v3;

            float3 points[] = { qlights[i].tri1.v1, qlights[i].tri1.v2, qlights[i].tri2.v2, qlights[i].tri1.v3 };

            float3 p1 = points[0]; float3 p2 = points[1]; float3 p3 = points[2]; float3 p4 = points[3];
            float theta_1 = acosf(dot(normalize(p1 - hitPt), normalize(p2 - hitPt)));
            float theta_2 = acosf(dot(normalize(p2 - hitPt), normalize(p3 - hitPt)));
            float theta_3 = acosf(dot(normalize(p3 - hitPt), normalize(p4 - hitPt)));
            float theta_4 = acosf(dot(normalize(p4 - hitPt), normalize(p1 - hitPt)));

            float3 gamma_1 = normalize(cross((p1 - hitPt), (p2 - hitPt)));
            float3 gamma_2 = normalize(cross((p2 - hitPt), (p3 - hitPt)));
            float3 gamma_3 = normalize(cross((p3 - hitPt), (p4 - hitPt)));
            float3 gamma_4 = normalize(cross((p4 - hitPt), (p1 - hitPt)));

            float3 irradiance_vec = 0.5f * ((theta_1 * gamma_1) +
                (theta_2 * gamma_2) + (theta_3 * gamma_3) + (theta_4 * gamma_4));

            float3 dir_radiance = f_brdf * qlights[i].color * dot(irradiance_vec, hitPtNormal);
            result += dir_radiance;
        }
    }
    else { // light_samples > 0
        for (int k = 0; k < qlights.size(); ++k) {
            float3 sampled_result = make_float3(.0f);
            // Compute direct lighting equation for w_i_k ray, for k = 1 to N*N
            float3 a = qlights[k].tri1.v1;
            float3 b = qlights[k].tri1.v2;
            float3 c = qlights[k].tri2.v2;
            float3 d = qlights[k].tri1.v3;

            //float3 f_brdf_1 = mv.diffuse / M_PIf;// brdf function 
            //float3 hitPt = attrib.intersection;
            //float3 hitPtNormal = attrib.normal;

            //float3 points[] = { qlights[k].tri1.v1, qlights[k].tri1.v2, qlights[k].tri2.v2, qlights[k].tri1.v3 };

            //float3 p1 = points[0]; float3 p2 = points[1]; float3 p3 = points[2]; float3 p4 = points[3];
            //float theta_1 = acosf(dot(normalize(p1 - hitPt), normalize(p2 - hitPt)));
            //float theta_2 = acosf(dot(normalize(p2 - hitPt), normalize(p3 - hitPt)));
            //float theta_3 = acosf(dot(normalize(p3 - hitPt), normalize(p4 - hitPt)));
            //float theta_4 = acosf(dot(normalize(p4 - hitPt), normalize(p1 - hitPt)));

            //float3 gamma_1 = normalize(cross((p1 - hitPt), (p2 - hitPt)));
            //float3 gamma_2 = normalize(cross((p2 - hitPt), (p3 - hitPt)));
            //float3 gamma_3 = normalize(cross((p3 - hitPt), (p4 - hitPt)));
            //float3 gamma_4 = normalize(cross((p4 - hitPt), (p1 - hitPt)));

            //float3 irradiance_vec = 0.5f * ((theta_1 * gamma_1) + 
            //    (theta_2 * gamma_2) + (theta_3 * gamma_3) + (theta_4 * gamma_4));

            //float3 dir_radiance = f_brdf_1 * qlights[k].color * dot(irradiance_vec, hitPtNormal);
            //sampled_result += dir_radiance;

            float3 ac = c - a;
            float3 ab = b - a;
            float area = length(cross(ab, ac));
            int root_light_samples = (int)sqrtf(light_samples);
            //rtPrintf("root of light samples and stratify: %d %d\n", root_light_samples, light_stratify);
            // check if stratify or random sampling
                // double for loop here 
            for (int i = 0; i < root_light_samples; ++i) {
                for (int j = 0; j < root_light_samples; ++j) {
                    // generate random float vals u1 and u2
                    float u1 = rnd(payload.seed);
                    float u2 = rnd(payload.seed);
                    //rtPrintf("%f %f \n", u1, u2);
                    float3 sampled_light_pos;
                    if (light_stratify) {
                        sampled_light_pos = a + ((j + u1) * (ab / (float)root_light_samples)) +
                            ((i + u2) * (ac / (float)root_light_samples));
                    }
                    else {
                        sampled_light_pos = a + u1 * ab + u2 * ac;
                    }
                    float3 shadow_ray_origin = attrib.intersection /*+ attrib.normal * cf.epsilon*/;
                    float3 shadow_ray_dir = normalize(sampled_light_pos - shadow_ray_origin);
                    float light_dist = length(sampled_light_pos - shadow_ray_origin);
                    Ray shadow_ray = make_Ray(shadow_ray_origin, shadow_ray_dir, 1, cf.epsilon, light_dist - cf.epsilon);

                    ShadowPayload shadow_payload;
                    shadow_payload.isVisible = true;
                    rtTrace(root, shadow_ray, shadow_payload);

                    //rtPrintf("%d", shadow_payload.isVisible);

                    if (shadow_payload.isVisible) {
                        // rendering equation here: 
                        //float3 w_i = sampled_light_pos;
                        float3 f_brdf = (mv.diffuse / M_PIf) +
                            (mv.specular * ((mv.shininess + 2.0f) / (2.0f * M_PIf)) *
                                powf(fmaxf(dot(normalize(reflect(-attrib.wo, attrib.normal)), normalize(sampled_light_pos - shadow_ray_origin)), .0f), mv.shininess));

                        float3 x_prime = sampled_light_pos;
                        float3 x = shadow_ray_origin;
                        float3 n = attrib.normal;
                        //float3 n_light = normalize(qlights[k].tri1.normal);
                        float3 n_light = normalize(cross(ab, ac));
                        //n_light = dot(n_light, normalize(x_prime - x)) > .0f ? n_light : -n_light;

                        float R = length(x - x_prime);

                        // note: normal should point AWAY from the hitpoint, i.e. dot(n_light, x - x_prime) < 0
                        float G = (1.0f / powf(R, 2.0f)) * fmaxf(dot(n, normalize(x_prime - x)), .0f) *
                            (fmaxf(dot(n_light, normalize(x_prime - x)), .0f));

                        sampled_result += f_brdf * G;
                    }

                }
            }
            result += qlights[k].color * sampled_result * (area / (float)light_samples);
        }
    }
    //}

    //result += sampled_result;



    // Compute the final radiance
    payload.radiance = result * payload.throughput;

    // Calculate reflection
    //if (length(mv.specular) > 0)
    //{
    //    // Set origin and dir for tracing the reflection ray
    //    payload.origin = attrib.intersection;
    //    payload.dir = reflect(-attrib.wo, attrib.normal); // mirror reflection

    //    payload.depth++;
    //    payload.throughput *= mv.specular;
    //}
    //else
    //{
    //    payload.done = true;
    //}
    payload.done = true;
}